/*
    -- KBLAS (version 1.0) --
       Ahmad Abdelfattah, Center of Extreme Computing
	   Hatem Ltaief, Supercomputing Laboratory
	   David Keyes, Center of Extreme Computing
	   King Abdullah University of Science and Technology (KAUST)
       June 2013
	   KBLAS is a subset of BLAS routines highly optimized for NVIDIA GPUs 
*/
/**
	-- Center of Extreme Computing and Supercomputing Laboratory
	-- Division of Applied Mathematics and Computational Science
	-- King Abdullah University of Science and Technology
	-- (C) Copyright 2013

	Redistribution  and  use  in  source and binary forms, with or without
	modification,  are  permitted  provided  that the following conditions
	are met:

	*	Redistributions  of  source  code  must  retain  the above copyright
		notice,  this  list  of  conditions  and  the  following  disclaimer.
	* 	Redistributions  in  binary  form must reproduce the above copyright
		notice,  this list of conditions and the following disclaimer in the
		documentation  and/or other materials provided with the distribution.
	* 	Neither  the  name of the University of Tennessee, Knoxville nor the
		names of its contributors may be used to endorse or promote products
		derived from this software without specific prior written permission.

	THIS  SOFTWARE  IS  PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
	''AS IS''  AND  ANY  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
	LIMITED  TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
	A  PARTICULAR  PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
	HOLDERS OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
	SPECIAL,  EXEMPLARY,  OR  CONSEQUENTIAL  DAMAGES  (INCLUDING,  BUT NOT
	LIMITED  TO,  PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
	DATA,  OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
	THEORY  OF  LIABILITY,  WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
	(INCLUDING  NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
	OF  THIS  SOFTWARE,  EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
**/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include "gemv2_core.cuh"

#if(SM >= 30)

#define cgemvn_nb               (32)
#define cgemvn_ntcol            (4)
#define cgemvn_ept              (2)
#define cgemvn_width    (cgemvn_ntcol*cgemvn_ept)
#define cgemvn_by               (16)

#define cgemvt_nb               (32)
#define cgemvt_ntcol            (2)
#define cgemvt_ept              (4)
#define cgemvt_width    (cgemvt_ntcol*cgemvt_ept)
#define cgemvt_by               (8)

#else

#define cgemvn_nb               (64)
#define cgemvn_ntcol    		(8)
#define cgemvn_ept              (2)
#define cgemvn_width    (cgemvn_ntcol*cgemvn_ept)
#define cgemvn_by               (1)

#define cgemvt_nb               (64)
#define cgemvt_ntcol    		(8)
#define cgemvt_ept              (2)
#define cgemvt_width    (cgemvt_ntcol*cgemvt_ept)
#define cgemvt_by               (1)
#endif


extern "C"
int kblas_cscal_async(int n, hipFloatComplex alpha, hipFloatComplex *x, int incx, hipStream_t stream);

  
int kblas_cgemv2_driver(	char trans, int rows, int cols,
						hipFloatComplex alpha, hipFloatComplex *dA, int lda, 
						hipFloatComplex *dX, int incx, 
						hipFloatComplex  beta, hipFloatComplex *dY, int incy,
						hipStream_t stream)
{	
	if(trans == 'n' || trans == 'N')
	{
		// scaling with beta
		kblas_cscal_async(rows, beta, dY, incy, stream);
		
		int mod_r = rows % cgemvn_nb;
		int mod_c = cols % cgemvn_width;	
		
		int blocks = rows/cgemvn_nb;
		if(mod_r != 0) blocks += 1;
		
		const int thread_x = cgemvn_nb;
		const int thread_y = cgemvn_ntcol; 
		const int ept = cgemvn_ept;
		
		int threshold = mod_c / ept; 
		int ept_ = mod_c % ept;
		dim3 dimBlock(thread_x, thread_y);
		dim3 dimGrid(blocks, cgemvn_by);
		switch(ept_)
		{
			case 0: gemvn<hipFloatComplex, cgemvn_nb, cgemvn_ntcol, ept, cgemvn_width, 0><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold); break;
			case 1: gemvn<hipFloatComplex, cgemvn_nb, cgemvn_ntcol, ept, cgemvn_width, 1><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold); break;
			case 2: gemvn<hipFloatComplex, cgemvn_nb, cgemvn_ntcol, ept, cgemvn_width, 2><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold); break;
			case 3: gemvn<hipFloatComplex, cgemvn_nb, cgemvn_ntcol, ept, cgemvn_width, 3><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold); break;
			case 4: gemvn<hipFloatComplex, cgemvn_nb, cgemvn_ntcol, ept, cgemvn_width, 4><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold); break;
			case 5: gemvn<hipFloatComplex, cgemvn_nb, cgemvn_ntcol, ept, cgemvn_width, 5><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold); break;
			case 6: gemvn<hipFloatComplex, cgemvn_nb, cgemvn_ntcol, ept, cgemvn_width, 6><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold); break;
			case 7: gemvn<hipFloatComplex, cgemvn_nb, cgemvn_ntcol, ept, cgemvn_width, 7><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold); break;
			case 8: gemvn<hipFloatComplex, cgemvn_nb, cgemvn_ntcol, ept, cgemvn_width, 8><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold); break;
			default: printf("irregular part %d is not supported, please extend the case statement of cgemv\n", ept_); exit(1);
		}
	}	// end of non-transpose case
	else if(trans == 't' || trans == 'T' || trans == 'c' || trans == 'C')
	{
		// scaling with beta
		kblas_cscal_async(cols, beta, dY, incy, stream);
		
		int mod_r = rows % cgemvt_nb;
		int mod_c = cols % cgemvt_width;
		
		int blocks = cols/cgemvt_width;
		if(mod_c != 0) blocks += 1;
		
		const int thread_x = cgemvt_nb;
		const int thread_y = cgemvt_ntcol;
		const int ept = cgemvt_ept;
		
		int threshold = mod_c / ept;
		int ept_ = mod_c % ept;
		
		dim3 dimBlock(thread_x, thread_y);
		dim3 dimGrid(blocks, cgemvt_by);
		
		int conj;
		if(trans == 'c' || trans == 'C')conj = 1;
		else conj = 0;
		//printf("modr = %d, modc = %d, threshold = %d, ept_ = %d \n", mod_r, mod_c, threshold, ept_);
		switch(ept_)
		{
			case 0: gemvt<hipFloatComplex, cgemvt_nb, cgemvt_ntcol, ept, cgemvt_width, 0><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, conj); break;
			case 1: gemvt<hipFloatComplex, cgemvt_nb, cgemvt_ntcol, ept, cgemvt_width, 1><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, conj); break;
			case 2: gemvt<hipFloatComplex, cgemvt_nb, cgemvt_ntcol, ept, cgemvt_width, 2><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, conj); break;
			case 3: gemvt<hipFloatComplex, cgemvt_nb, cgemvt_ntcol, ept, cgemvt_width, 3><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, conj); break;
			case 4: gemvt<hipFloatComplex, cgemvt_nb, cgemvt_ntcol, ept, cgemvt_width, 4><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, conj); break;
			case 5: gemvt<hipFloatComplex, cgemvt_nb, cgemvt_ntcol, ept, cgemvt_width, 5><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, conj); break;
			case 6: gemvt<hipFloatComplex, cgemvt_nb, cgemvt_ntcol, ept, cgemvt_width, 6><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, conj); break;
			case 7: gemvt<hipFloatComplex, cgemvt_nb, cgemvt_ntcol, ept, cgemvt_width, 7><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, conj); break;
			case 8: gemvt<hipFloatComplex, cgemvt_nb, cgemvt_ntcol, ept, cgemvt_width, 8><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, conj); break;
			default: printf("irregular part %d is not supported, please extend the case statement of cgemv\n", ept_); exit(1);
		}
	}
	else
	{	
		printf("CGEMV error: Unrecognized transpose mode %c \n", trans);
		return -1;
	}
	
	return 0;
}

extern "C"
int kblas_cgemv2(char trans, int rows, int cols,
				hipFloatComplex alpha, hipFloatComplex *dA, int lda, 
				hipFloatComplex *dX, int incx, 
				hipFloatComplex  beta, hipFloatComplex *dY, int incy)
{
	return kblas_cgemv2_driver(	trans, rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, 0);
}

extern "C"
int kblas_cgemv2_async(	char trans, int rows, int cols,
						hipFloatComplex alpha, hipFloatComplex *dA, int lda, 
						hipFloatComplex *dX, int incx, 
						hipFloatComplex  beta, hipFloatComplex *dY, int incy,
						hipStream_t stream)
{
	return kblas_cgemv2_driver(	trans, rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, stream);
}
