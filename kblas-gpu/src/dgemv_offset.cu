/*
    -- KBLAS (version 1.0) --
       Ahmad Abdelfattah, Center of Extreme Computing
	   Hatem Ltaief, Supercomputing Laboratory
	   David Keyes, Center of Extreme Computing
	   King Abdullah University of Science and Technology (KAUST)
       June 2013
	   KBLAS is a subset of BLAS routines highly optimized for NVIDIA GPUs 
*/
/**
	-- Center of Extreme Computing and Supercomputing Laboratory
	-- Division of Applied Mathematics and Computational Science
	-- King Abdullah University of Science and Technology
	-- (C) Copyright 2013

	Redistribution  and  use  in  source and binary forms, with or without
	modification,  are  permitted  provided  that the following conditions
	are met:

	*	Redistributions  of  source  code  must  retain  the above copyright
		notice,  this  list  of  conditions  and  the  following  disclaimer.
	* 	Redistributions  in  binary  form must reproduce the above copyright
		notice,  this list of conditions and the following disclaimer in the
		documentation  and/or other materials provided with the distribution.
	* 	Neither  the  name of the University of Tennessee, Knoxville nor the
		names of its contributors may be used to endorse or promote products
		derived from this software without specific prior written permission.

	THIS  SOFTWARE  IS  PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
	''AS IS''  AND  ANY  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
	LIMITED  TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
	A  PARTICULAR  PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
	HOLDERS OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
	SPECIAL,  EXEMPLARY,  OR  CONSEQUENTIAL  DAMAGES  (INCLUDING,  BUT NOT
	LIMITED  TO,  PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
	DATA,  OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
	THEORY  OF  LIABILITY,  WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
	(INCLUDING  NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
	OF  THIS  SOFTWARE,  EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
**/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include "gemv_offset_core.cuh"

#if(SM >= 30)

#define dgemvn_offset_bs		(32)
#define dgemvn_offset_ty		(8)
#define dgemvn_offset_by		(4)

#define dgemvt_offset_bs		(32)
#define dgemvt_offset_ty		(8)
#define dgemvt_offset_by		(4)

#else

#define dgemvn_offset_bs		(64)
#define dgemvn_offset_ty		(8)
#define dgemvn_offset_by		(2)

#define dgemvt_offset_bs		(64)
#define dgemvt_offset_ty		(8)
#define dgemvt_offset_by		(2)
#endif


extern "C"
int kblas_dscal_async(int n, double alpha, double *x, int incx, hipStream_t stream);

int kblas_dgemv_offset_driver( char trans, int rows, int cols,
						        double alpha, double *dA, int lda, 
						        double *dX, int incx, 
						        double  beta, double *dY, int incy,
						        int offset_r, int offset_c, 
						        hipStream_t stream = 0)
{
	if(trans == 'n' || trans == 'N')
	{
		
		//**** Config parameters
		const int thread_x = dgemvn_offset_bs;
		const int thread_y = dgemvn_offset_ty; 
		const int elements_per_thread = thread_x/(2*thread_y);
		const int grid_y_n = dgemvn_offset_by;
		//*************************
		
		/** offset necessary calculation **/
		int offset_r_ = offset_r % dgemvn_offset_bs;
		int offset_c_ = offset_c % dgemvn_offset_bs;
		int total_blocks_skipped_r = offset_r / dgemvn_offset_bs; 
		int total_blocks_skipped_c = offset_c / dgemvn_offset_bs; 
		int my_skipped_blocks_r = total_blocks_skipped_r;
		int my_skipped_blocks_c = total_blocks_skipped_c/ngpus; 
		if(gpu_gid < (total_blocks_skipped_c%ngpus)) my_skipped_blocks_c += 1;
		int ref_gpu = total_blocks_skipped_c%ngpus; 
		int new_gpu_gid = (gpu_gid - ref_gpu + ngpus) % ngpus;
		// Advance pointers accordingly
		dA += my_skipped_blocks_c * dgemvn_offset_bs * lda;
		dA += my_skipped_blocks_r * dgemvn_offset_bs; 
		dX += my_skipped_blocks_c * dgemvn_offset_bs * incx;
		dY += my_skipped_blocks_r * dgemvn_offset_bs * incy;
		rows -= my_skipped_blocks_r * dgemvn_offset_bs;
		cols -= my_skipped_blocks_c * dgemvn_offset_bs;
		/** end offset necessary calculation **/
		
		int nstripes = (cols/dgemvn_offset_bs) + ((cols%dgemvn_offset_bs) != 0);
		
		// scaling with beta
		//if(gpu_gid == 0)hipblasSscal(rows-offset_, beta, dY+(offset_*incy), incy);
		if(gpu_gid == 0)kblas_dscal_async(rows-offset_r_, beta, dY+(offset_r_*incy), incy, stream);
		
		int cols_ = dgemvn_offset_bs * ( (cols/dgemvn_offset_bs)/ngpus );
		if(new_gpu_gid < (cols/dgemvn_offset_bs)%ngpus) cols_ += dgemvn_offset_bs;
		if(new_gpu_gid == (cols/dgemvn_offset_bs)%ngpus) cols_ += cols%dgemvn_offset_bs;
		
		int mod_r = rows % dgemvn_offset_bs;
		int mod_c = cols_ % dgemvn_offset_bs;
		
		if(mod_r == 0 && mod_c == 0)
		{
			// special case
			int blocks = rows/dgemvn_offset_bs;
			dim3 dimBlock(thread_x, thread_y);
			dim3 dimGrid(blocks, grid_y_n);
			if(blocks == 0) return 0;
			gemvn_special_offset<double, dgemvn_offset_bs, dgemvn_offset_bs, dgemvn_offset_ty, elements_per_thread>
									<<<dimGrid, dimBlock, 0, stream>>>
									(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, nstripes, offset_r_, offset_c_);
		}
		else
		{
			// generic case for columns only
			const int irregular_cols = mod_c % elements_per_thread;
			
			int blocks = (rows/dgemvn_offset_bs) + (mod_r != 0);
			if(mod_r == 0)blocks += 1;	// dummy thread block, will return immediately if mod_r == 0
			
			dim3 dimBlock(thread_x, thread_y);
			dim3 dimGrid(blocks, grid_y_n);
			if(blocks == 0) return 0;
			
			switch(irregular_cols)
			{
				/**
				 * The kernel for irregular dimensions has an extra template parameter.
				 * This parameter must be among the values listed in the switch-case statement below.
				 * The possible values are in the range 0 - (elements_per_thread-1)
				 * Make sure these values are updated whenever you change the configuration parameters.  
				**/	
				case  0: gemvn_generic_offset<double, dgemvn_offset_bs, dgemvn_offset_bs, dgemvn_offset_ty, elements_per_thread,  0><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_); break;
				case  1: gemvn_generic_offset<double, dgemvn_offset_bs, dgemvn_offset_bs, dgemvn_offset_ty, elements_per_thread,  1><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_); break;
				case  2: gemvn_generic_offset<double, dgemvn_offset_bs, dgemvn_offset_bs, dgemvn_offset_ty, elements_per_thread,  2><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_); break;
				case  3: gemvn_generic_offset<double, dgemvn_offset_bs, dgemvn_offset_bs, dgemvn_offset_ty, elements_per_thread,  3><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_); break;
				case  4: gemvn_generic_offset<double, dgemvn_offset_bs, dgemvn_offset_bs, dgemvn_offset_ty, elements_per_thread,  4><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_); break;
				case  5: gemvn_generic_offset<double, dgemvn_offset_bs, dgemvn_offset_bs, dgemvn_offset_ty, elements_per_thread,  5><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_); break;
				case  6: gemvn_generic_offset<double, dgemvn_offset_bs, dgemvn_offset_bs, dgemvn_offset_ty, elements_per_thread,  6><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_); break;
				case  7: gemvn_generic_offset<double, dgemvn_offset_bs, dgemvn_offset_bs, dgemvn_offset_ty, elements_per_thread,  7><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_); break;
				case  8: gemvn_generic_offset<double, dgemvn_offset_bs, dgemvn_offset_bs, dgemvn_offset_ty, elements_per_thread,  8><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_); break;
				case  9: gemvn_generic_offset<double, dgemvn_offset_bs, dgemvn_offset_bs, dgemvn_offset_ty, elements_per_thread,  9><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_); break;
				case 10: gemvn_generic_offset<double, dgemvn_offset_bs, dgemvn_offset_bs, dgemvn_offset_ty, elements_per_thread, 10><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_); break;
				case 11: gemvn_generic_offset<double, dgemvn_offset_bs, dgemvn_offset_bs, dgemvn_offset_ty, elements_per_thread, 11><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_); break;
				case 12: gemvn_generic_offset<double, dgemvn_offset_bs, dgemvn_offset_bs, dgemvn_offset_ty, elements_per_thread, 12><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_); break;
				case 13: gemvn_generic_offset<double, dgemvn_offset_bs, dgemvn_offset_bs, dgemvn_offset_ty, elements_per_thread, 13><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_); break;
				case 14: gemvn_generic_offset<double, dgemvn_offset_bs, dgemvn_offset_bs, dgemvn_offset_ty, elements_per_thread, 14><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_); break;
				case 15: gemvn_generic_offset<double, dgemvn_offset_bs, dgemvn_offset_bs, dgemvn_offset_ty, elements_per_thread, 15><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_); break;
				default: printf("DGEMV-N error: improper template parameter. Please read the inline documentation for this function. \n"); return -1;	
			}
		}
	}	// end of non-transpose case
	else if (trans == 't' || trans == 'T' || trans == 'c' || trans == 'C')
	{
		int conj;
		if(trans == 'c' || trans == 'C') conj = 1;
		else conj = 0;
		//**** Config parameters
		const int thread_x = dgemvt_offset_bs;
		const int thread_y = dgemvt_offset_ty; 
		const int elements_per_thread = thread_x/(2*thread_y);
		const int grid_y_t = dgemvt_offset_by;
		//*************************
		
		/** offset necessary calculation **/
		int offset_r_ = offset_r % dgemvt_offset_bs;
		int offset_c_ = offset_c % dgemvt_offset_bs;
		int total_blocks_skipped_r = offset_r / dgemvt_offset_bs; 
		int total_blocks_skipped_c = offset_c / dgemvt_offset_bs; 
		int my_skipped_blocks_r = total_blocks_skipped_r; 
		int my_skipped_blocks_c = total_blocks_skipped_c/ngpus; 
		if(gpu_gid < (total_blocks_skipped_c%ngpus)) my_skipped_blocks_c += 1;
		int ref_gpu = total_blocks_skipped_c%ngpus; 
		int new_gpu_gid = (gpu_gid - ref_gpu + ngpus) % ngpus;
		
		//if(new_gpu_gid != 3){return 0;}
		// Advance pointers accordingly
		dA += my_skipped_blocks_c * dgemvt_offset_bs * lda;
		dA += my_skipped_blocks_r * dgemvt_offset_bs; 
		dX += my_skipped_blocks_r * dgemvt_offset_bs * incx;
		dY += my_skipped_blocks_c * dgemvt_offset_bs * incy;
		rows -= my_skipped_blocks_r * dgemvt_offset_bs;
		cols -= my_skipped_blocks_c * dgemvt_offset_bs;
		/** end offset necessary calculation **/
		
		int nstripes = (cols/dgemvt_offset_bs) + ((cols%dgemvt_offset_bs) != 0);
		
		// scaling with beta
		//if(gpu_gid == 0)hipblasSscal(cols-offset_, beta, dY+(offset_*incy), incy);
		if(gpu_gid == 0)kblas_dscal_async(cols-offset_c_, beta, dY+(offset_c_*incy), incy, stream);
		
		int cols_ = dgemvt_offset_bs * ( (cols/dgemvt_offset_bs)/ngpus );
		if(new_gpu_gid < (cols/dgemvt_offset_bs)%ngpus) cols_ += dgemvt_offset_bs;
		if(new_gpu_gid == (cols/dgemvt_offset_bs)%ngpus) cols_ += cols%dgemvt_offset_bs;
		
		int mod_r = rows % dgemvt_offset_bs;
		int mod_c = cols_ % dgemvt_offset_bs;
		
		if(mod_r == 0 && mod_c == 0)
		{	
			int blocks = cols_/dgemvt_offset_bs;
			dim3 dimBlock(thread_x, thread_y);
			dim3 dimGrid(blocks, grid_y_t);
			if(blocks == 0) return 0;
			gemvt_special_offset<double, dgemvt_offset_bs, thread_x, thread_y, elements_per_thread><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, nstripes, offset_r_, offset_c_, conj); 
			
		}
		else
		{
			const int irregular_cols = mod_c % elements_per_thread;
			int blocks = cols_/dgemvt_offset_bs + (mod_c != 0);
			int gpu_last = (nstripes+ngpus-1)%ngpus;
			if(mod_c == 0 && new_gpu_gid == gpu_last) blocks += 1; // dummy thread block, will return if mod_c == 0
			dim3 dimBlock(thread_x, thread_y);
			dim3 dimGrid(blocks, grid_y_t);
			if(blocks == 0) return 0;
			
			switch(irregular_cols)
			{
				/**
				  * The kernel for irregular dimensions has an extra template parameter.
				  * This parameter must be among the values listed in the switch-case statement below.
				  * The possible values are in the range 0 - (elements_per_thread-1)
				  * Make sure these values are updated whenever you change the configuration parameters.  
				**/	
				case  0: gemvt_generic_offset<double, dgemvt_offset_bs, thread_x, thread_y, elements_per_thread,  0><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_, conj); break;
				case  1: gemvt_generic_offset<double, dgemvt_offset_bs, thread_x, thread_y, elements_per_thread,  1><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_, conj); break;
				case  2: gemvt_generic_offset<double, dgemvt_offset_bs, thread_x, thread_y, elements_per_thread,  2><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_, conj); break;
				case  3: gemvt_generic_offset<double, dgemvt_offset_bs, thread_x, thread_y, elements_per_thread,  3><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_, conj); break;
				case  4: gemvt_generic_offset<double, dgemvt_offset_bs, thread_x, thread_y, elements_per_thread,  4><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_, conj); break;
				case  5: gemvt_generic_offset<double, dgemvt_offset_bs, thread_x, thread_y, elements_per_thread,  5><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_, conj); break;
				case  6: gemvt_generic_offset<double, dgemvt_offset_bs, thread_x, thread_y, elements_per_thread,  6><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_, conj); break;
				case  7: gemvt_generic_offset<double, dgemvt_offset_bs, thread_x, thread_y, elements_per_thread,  7><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_, conj); break;
				case  8: gemvt_generic_offset<double, dgemvt_offset_bs, thread_x, thread_y, elements_per_thread,  8><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_, conj); break;
				case  9: gemvt_generic_offset<double, dgemvt_offset_bs, thread_x, thread_y, elements_per_thread,  9><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_, conj); break;
				case 10: gemvt_generic_offset<double, dgemvt_offset_bs, thread_x, thread_y, elements_per_thread, 10><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_, conj); break;
				case 11: gemvt_generic_offset<double, dgemvt_offset_bs, thread_x, thread_y, elements_per_thread, 11><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_, conj); break;
				case 12: gemvt_generic_offset<double, dgemvt_offset_bs, thread_x, thread_y, elements_per_thread, 12><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_, conj); break;
				case 13: gemvt_generic_offset<double, dgemvt_offset_bs, thread_x, thread_y, elements_per_thread, 13><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_, conj); break;
				case 14: gemvt_generic_offset<double, dgemvt_offset_bs, thread_x, thread_y, elements_per_thread, 14><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_, conj); break;
				case 15: gemvt_generic_offset<double, dgemvt_offset_bs, thread_x, thread_y, elements_per_thread, 15><<<dimGrid, dimBlock, 0, stream>>>(rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, nstripes, offset_r_, offset_c_, conj); break;
				default: printf("DGEMV-T error: improper template parameter. Please read the inline documentation for this function. \n"); return -1;
			}
		}
	}
	else 
	{
		printf("DGEMV error: Unrecognized transpose mode %c \n", trans);
		return -1;
	}
	
	return 0;
}
/***********************************************************************************/
extern "C"
int kblas_dgemv_offset( char trans, int rows, int cols,
						double alpha, double *dA, int lda, 
						double *dX, int incx, 
						double beta, double *dY, int incy, 
						int offset_r, int offset_c)
{
	return kblas_dgemv_offset_driver( trans, rows, cols, 
	                                alpha, dA, lda, 
	                                dX, incx, 
	                                beta, dY, incy, 
	                                offset_r, offset_c);
}

/*************************************************************************************/
extern "C"
int kblas_dgemv_offset_async( char trans, int rows, int cols,
							double alpha, double *dA, int lda, 
							double *dX, int incx, 
							double  beta, double *dY, int incy, 
							int offset_r, int offset_c,
							hipStream_t stream)
{
	return kblas_dgemv_offset_driver(  trans, rows, cols, 
	                            alpha, dA, lda, 
	                            dX, incx, 
	                            beta, dY, incy, offset_r, offset_c, 
	                            stream);
}
/*************************************************************************************/
