 /**
  - -* (C) Copyright 2013 King Abdullah University of Science and Technology
  Authors:
  Ali Charara (ali.charara@kaust.edu.sa)
  David Keyes (david.keyes@kaust.edu.sa)
  Hatem Ltaief (hatem.ltaief@kaust.edu.sa)
  
  Redistribution  and  use  in  source and binary forms, with or without
  modification,  are  permitted  provided  that the following conditions
  are met:
  
  * Redistributions  of  source  code  must  retain  the above copyright
  * notice,  this  list  of  conditions  and  the  following  disclaimer.
  * Redistributions  in  binary  form must reproduce the above copyright
  * notice,  this list of conditions and the following disclaimer in the
  * documentation  and/or other materials provided with the distribution.
  * Neither  the  name of the King Abdullah University of Science and
  * Technology nor the names of its contributors may be used to endorse
  * or promote products derived from this software without specific prior
  * written permission.
  * 
  THIS  SOFTWARE  IS  PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
  ``AS IS''  AND  ANY  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
  LIMITED  TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
  A  PARTICULAR  PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
  HOLDERS OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
  SPECIAL,  EXEMPLARY,  OR  CONSEQUENTIAL  DAMAGES  (INCLUDING,  BUT NOT
  LIMITED  TO,  PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
  DATA,  OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
  THEORY  OF  LIABILITY,  WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
  (INCLUDING  NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
  OF  THIS  SOFTWARE,  EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
  **/
#include <stdlib.h>
#include <stdio.h>
#include <set>
#include <hipblas.h>
#include "kblas.h"
#include "operators.h"
#include "Xtr_common.ch"

//==============================================================================================
extern "C"{
const char* cublasGetErrorString( hipblasStatus_t error )
{
  switch( error ) {
    case HIPBLAS_STATUS_SUCCESS:
      return "success";
      
    case HIPBLAS_STATUS_NOT_INITIALIZED:
      return "not initialized";
      
    case HIPBLAS_STATUS_ALLOC_FAILED:
      return "out of memory";
      
    case HIPBLAS_STATUS_INVALID_VALUE:
      return "invalid value";
      
    case HIPBLAS_STATUS_ARCH_MISMATCH:
      return "architecture mismatch";
      
    case HIPBLAS_STATUS_MAPPING_ERROR:
      return "memory mapping error";
      
    case HIPBLAS_STATUS_EXECUTION_FAILED:
      return "execution failed";
      
    case HIPBLAS_STATUS_INTERNAL_ERROR:
      return "internal error";

    case HIPBLAS_STATUS_NOT_SUPPORTED:
      return "Not Supported";

    case HIPBLAS_STATUS_UNKNOWN:
      return "License Error";

    default:
      return "unknown CUBLAS error code";
  }
}
}

// ----------------------------------------
// C++ function is overloaded for different error types,
// which depends on error types being enums to be differentiable.
//inline
int _kblas_error( hipError_t err, const char* func, const char* file, int line )
{
  if ( err != hipSuccess ) {
    fprintf( stderr, "CUDA runtime error: %s (%d) in %s at %s:%d\n",
             hipGetErrorString( err ), err, func, file, line );
    return 0;
  }
  return 1;
}

// --------------------
//inline
int _kblas_error( hipblasStatus_t err, const char* func, const char* file, int line )
{
  if ( err != HIPBLAS_STATUS_SUCCESS ) {
    fprintf( stderr, "CUBLAS error: %s (%d) in %s at %s:%d\n",
             cublasGetErrorString( err ), err, func, file, line );
    return 0;
  }
  return 1;
}

//#define check_error( err ) \
//{if(!_kblas_error( (err), __func__, __FILE__, __LINE__ )) return 0;}

//==============================================================================================
bool REG_SIZE(int n){
  return ((n > 0) && !(n & (n - 1)));
}
int CLOSEST_REG_SIZE(int n){
  //TODO validate input
  if(n > 0){
    int res = 1;
    while (res < n){
      res = res << 1;
    }
    return res >> 1;
  }else{
    return 0;    
  }
}

//==============================================================================================

hipblasStatus_t cublasXgemm( hipblasHandle_t handle,
                            hipblasOperation_t transa, hipblasOperation_t transb,
                            int m, int n, int k,
                            const float *alpha, const float *A, int lda,
                                                const float *B, int ldb,
                            const float *beta,        float *C, int ldc){
  hipblasStatus_t status;
  check_error( status = hipblasSgemm(handle,
                                    transa, transb,
                                    m, n, k,
                                    alpha, A, lda,
                                           B, ldb,
                                    beta,  C, ldc), status);
  check_error( hipGetLastError(), HIPBLAS_STATUS_EXECUTION_FAILED );
  return HIPBLAS_STATUS_SUCCESS;
}

hipblasStatus_t cublasXgemm( hipblasHandle_t handle,
                            hipblasOperation_t transa, hipblasOperation_t transb,
                            int m, int n, int k,
                            const double *alpha, const double *A, int lda,
                                                 const double *B, int ldb,
                            const double *beta,        double *C, int ldc){
  hipblasStatus_t status;
  check_error( status = hipblasDgemm(handle,
                                    transa, transb,
                                    m, n, k,
                                    alpha, A, lda,
                                           B, ldb,
                                    beta,  C, ldc), status);
  check_error( hipGetLastError(), HIPBLAS_STATUS_EXECUTION_FAILED );
  return HIPBLAS_STATUS_SUCCESS;
}
hipblasStatus_t cublasXgemm(hipblasHandle_t handle,
                           hipblasOperation_t transa, hipblasOperation_t transb,
                           int m, int n, int k,
                           const hipComplex *alpha, const hipComplex *A, int lda,
                                                   const hipComplex *B, int ldb,
                           const hipComplex *beta,        hipComplex *C, int ldc){
  hipblasStatus_t status;
  check_error( status = hipblasCgemm(handle,
                                    transa, transb,
                                    m, n, k,
                                    alpha, A, lda,
                                           B, ldb,
                                    beta,  C, ldc), status);
  check_error( hipGetLastError(), HIPBLAS_STATUS_EXECUTION_FAILED );
  return HIPBLAS_STATUS_SUCCESS;
}
hipblasStatus_t cublasXgemm(hipblasHandle_t handle,
                           hipblasOperation_t transa, hipblasOperation_t transb,
                           int m, int n, int k,
                           const hipDoubleComplex *alpha, const hipDoubleComplex *A, int lda,
                                                         const hipDoubleComplex *B, int ldb,
                           const hipDoubleComplex *beta,        hipDoubleComplex *C, int ldc){
  hipblasStatus_t status;
  check_error( status = hipblasZgemm(handle,
                                    transa, transb,
                                    m, n, k,
                                    alpha, A, lda,
                                           B, ldb,
                                    beta,  C, ldc), status);
  check_error( hipGetLastError(), HIPBLAS_STATUS_EXECUTION_FAILED );
  return HIPBLAS_STATUS_SUCCESS;
}



