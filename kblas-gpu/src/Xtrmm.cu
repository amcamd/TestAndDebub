#include "hip/hip_runtime.h"
/**
  -- (C) Copyright 2013 King Abdullah University of Science and Technology
  Authors:
  Ali Charara (ali.charara@kaust.edu.sa)
  David Keyes (david.keyes@kaust.edu.sa)
  Hatem Ltaief (hatem.ltaief@kaust.edu.sa)
  
  Redistribution  and  use  in  source and binary forms, with or without
  modification,  are  permitted  provided  that the following conditions
  are met:
  
  * Redistributions  of  source  code  must  retain  the above copyright
  * notice,  this  list  of  conditions  and  the  following  disclaimer.
  * Redistributions  in  binary  form must reproduce the above copyright
  * notice,  this list of conditions and the following disclaimer in the
  * documentation  and/or other materials provided with the distribution.
  * Neither  the  name of the King Abdullah University of Science and
  * Technology nor the names of its contributors may be used to endorse
  * or promote products derived from this software without specific prior
  * written permission.
  * 
  THIS  SOFTWARE  IS  PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
  ``AS IS''  AND  ANY  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
  LIMITED  TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
  A  PARTICULAR  PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
  HOLDERS OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
  SPECIAL,  EXEMPLARY,  OR  CONSEQUENTIAL  DAMAGES  (INCLUDING,  BUT NOT
  LIMITED  TO,  PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
  DATA,  OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
  THEORY  OF  LIABILITY,  WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
  (INCLUDING  NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
  OF  THIS  SOFTWARE,  EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
**/
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "hipblas.h"
#include "kblas.h"
#include "Xtr_common.ch"
#include "operators.h"

//==============================================================================================

hipblasStatus_t cublasXtrmm(hipblasHandle_t handle,
                           hipblasSideMode_t side, hipblasFillMode_t uplo,
                           hipblasOperation_t trans, hipblasDiagType_t diag,
                           int m, int n,
                           const float *alpha,
                           const float *A, int lda,
                                 float *B, int ldb){
  hipblasStatus_t status;
  check_error( status = hipblasStrmm(handle,
                                    side, uplo, trans, diag,
                                    m, n,
                                    alpha, A, lda,
                                           B, ldb,
                                           B, ldb ), status);
  check_error( hipGetLastError(), HIPBLAS_STATUS_EXECUTION_FAILED );
  return HIPBLAS_STATUS_SUCCESS;
}
hipblasStatus_t cublasXtrmm(hipblasHandle_t handle,
                           hipblasSideMode_t side, hipblasFillMode_t uplo,
                           hipblasOperation_t trans, hipblasDiagType_t      diag,
                           int m, int n,
                           const double *alpha,
                           const double *A, int lda,
                                 double *B, int ldb){
  hipblasStatus_t status;
  check_error( status = hipblasDtrmm(handle,
                                    side, uplo, trans, diag,
                                    m, n,
                                    alpha, A, lda,
                                           B, ldb,
                                           B, ldb ), status);
  check_error( hipGetLastError(), HIPBLAS_STATUS_EXECUTION_FAILED );
  return HIPBLAS_STATUS_SUCCESS;
}
hipblasStatus_t cublasXtrmm (hipblasHandle_t handle,
                            hipblasSideMode_t side, hipblasFillMode_t uplo,
                            hipblasOperation_t trans, hipblasDiagType_t diag,
                            int m, int n,
                            const hipComplex *alpha,
                            const hipComplex *A, int lda,
                                  hipComplex *B, int ldb){
  hipblasStatus_t status;
  check_error( status = hipblasCtrmm(handle,
                                    side, uplo, trans, diag,
                                    m, n,
                                    alpha, A, lda,
                                           B, ldb,
                                           B, ldb ), status);
  check_error( hipGetLastError(), HIPBLAS_STATUS_EXECUTION_FAILED );
  return HIPBLAS_STATUS_SUCCESS;
}
hipblasStatus_t cublasXtrmm (hipblasHandle_t handle,
                            hipblasSideMode_t side, hipblasFillMode_t uplo,
                            hipblasOperation_t trans, hipblasDiagType_t diag,
                            int m, int n,
                            const hipDoubleComplex *alpha,
                            const hipDoubleComplex *A, int lda,
                                  hipDoubleComplex *B, int ldb){
  hipblasStatus_t status;
  check_error( status = hipblasZtrmm(handle,
                                    side, uplo, trans, diag,
                                    m, n,
                                    alpha, A, lda,
                                           B, ldb,
                                           B, ldb ), status);
  check_error( hipGetLastError(), HIPBLAS_STATUS_EXECUTION_FAILED );
  return HIPBLAS_STATUS_SUCCESS;
}


//==============================================================================================
#define WARP 32
#define WARP1 33
#define tx threadIdx.x
#define ty threadIdx.y
//==============================================================================================
int kblas_trmm_ib_custom = 128;
int kblas_trmm_ib_cublas = 128;
int kblas_trmm_ib_data = 512;
bool kblas_trmm_use_custom = 0;
//#define SIMPLE_SIZE_CUSTOM(n) ( ((n)<32) || ((n) % 32 == 0 && (n) <= kblas_trmm_ib_custom) )
#define SIMPLE_SIZE(n) ( ((n) < WARP) || ( ((n) % WARP == 0) && ( (n) <= kblas_trmm_ib_cublas ) ) )
#define SIMPLE_SIZE_DATA(n) ( (n) <= kblas_trmm_ib_data )

//shuffle intrinsic is not supported before KEPLER
#if (SM >= 30)
//==============================================================================================
template<typename T, int WARPS_PER_BLOCK, int B_COLS_PER_WARP, bool LOWER, bool TRANS, bool CONJG>
__global__ void //__launch_bounds__(256)
trmm_mul32_L(int M, int N, T alpha, const T* __restrict__ A, int incA, T* B, int incB, int mb){
  
  const int A_COL_PER_WARP = WARP / WARPS_PER_BLOCK;
  const bool forward = (LOWER == TRANS);
  
  int txyw = tx + ty*WARP1/*, tyxw = ty + tx*WARP1*/, txyiA = tx + ty*incA, txyiB = tx + ty*incB;
  
  //setup shared memory
  __shared__ T sA[WARP * WARP1];//strided to avoid bank conflict
  T rB[B_COLS_PER_WARP], rBj[B_COLS_PER_WARP], s[B_COLS_PER_WARP], a[4], b[4], *sAA, *BB;
  int c, j, r, l, i, startB = 0, active_col;
  
  for(startB = 0; startB < N; startB += gridDim.x * WARPS_PER_BLOCK * B_COLS_PER_WARP)
  {

    if( (startB + blockIdx.x * WARPS_PER_BLOCK * B_COLS_PER_WARP) >= N) return;
    
    BB = B + (startB + blockIdx.x * WARPS_PER_BLOCK * B_COLS_PER_WARP) * incB;
    active_col = 0;//an inactive warp will still contribute to data fetching but not to computation
    
    #pragma unroll
    for(l = 0; l < B_COLS_PER_WARP; l++)
      active_col += ((startB + blockIdx.x * (WARPS_PER_BLOCK * B_COLS_PER_WARP) + ty + l * WARPS_PER_BLOCK) < N);

    for( c = (forward ? 0 : mb-1); (forward && (c < mb)) || (!forward && (c > -1)); c += (forward ? 1 : -1))
    {
      #pragma unroll
      for(l = 0; l < B_COLS_PER_WARP; l++)
        s[l] = make_zero<T>();
      //load A(c,c) from global to shared mem
      #pragma unroll
      for(l = 0; l < A_COL_PER_WARP; l++)
        sA[txyw + l * WARPS_PER_BLOCK * WARP1] = A[txyiA + WARP * c * (incA+1) + l * WARPS_PER_BLOCK * incA];
      
      //load B(c) into registers
      #pragma unroll
      for(l = 0; l < B_COLS_PER_WARP; l++)
        if(active_col > l)
          rB[l] = BB[txyiB + WARP * c + l * WARPS_PER_BLOCK * incB];
      
      /*__syncthreads();
      if(forward){
        #pragma unroll
        for(l = 0; l < A_COL_PER_WARP; l++)
          if(tx < (ty + l * WARPS_PER_BLOCK))
            sA[(ty + l * WARPS_PER_BLOCK) + WARP1 * tx] = sA[tx + WARP1 * (ty + l * WARPS_PER_BLOCK)];
      }*/
      __syncthreads();

      //perform trmm on shared mem
      if(active_col > 0){
        if(forward){
          #pragma unroll
          for(j = 0; j < WARP; j++){
            #pragma unroll
            for(l = 0; l < B_COLS_PER_WARP; l++)
                rBj[l] = shfl(rB[l], j);
            if(j >= tx){
              //a[0] = CONJG ? conjugate(sA[tx + j * WARP1]) : sA[tx + j * WARP1];
              a[0] = CONJG ? conjugate(sA[j + tx * WARP1]) : sA[j + tx * WARP1];//TODO
              #pragma unroll
              for(l = 0; l < B_COLS_PER_WARP; l++)
                  s[l] = FMA( a[0], rBj[l], s[l]);
            }
          }
        }else{
          /*#pragma unroll
          for(j = WARP-1; j > -1; j-=4){
            #pragma unroll
            for(i = 3; i > -1; i--){
              a[i] = ((j-i) > tx) ? make_zero<T>() : (CONJG ? conjugate(sA[tx + (j-i) * WARP1]) : sA[tx + (j-i) * WARP1]);
            }
            #pragma unroll
            for(l = 0; l < B_COLS_PER_WARP; l++){
              #pragma unroll
              for(i = 3; i > -1; i--)
                b[i] = shfl(rB[l], j-i);
              #pragma unroll
              for(i = 3; i > -1; i--)
                s[l] = FMA( a[i], b[i], s[l]);
            }
          }/*/
          #pragma unroll
          for(j = WARP-1; j > -1; j--){
            #pragma unroll
            for(l = 0; l < B_COLS_PER_WARP; l++)
              rBj[l] = shfl(rB[l], j);
            if(j <= tx){
              a[0] = CONJG ? conjugate(sA[tx + j * WARP1]) : sA[tx + j * WARP1];
              #pragma unroll
              for(l = 0; l < B_COLS_PER_WARP; l++)
                s[l] = FMA( a[0], rBj[l], s[l]);
            }
          }//*/
        }
      }
      __syncthreads();

      for(r = (forward ? c+1 : 0); (forward && (r < mb)) || (!forward && (r < c)); r++){
        #pragma unroll
        for(l = 0; l < A_COL_PER_WARP; l++){
          if(TRANS)//load A(r,c)
            //sA[tyxw + l * WARPS_PER_BLOCK] = A[txyiA + WARP * (r + c * incA) + l * WARPS_PER_BLOCK * incA];
            sA[txyw + l * WARPS_PER_BLOCK * WARP1] = A[txyiA + WARP * (r + c * incA) + l * WARPS_PER_BLOCK * incA];
          else//load A(c,r)
            sA[txyw + l * WARPS_PER_BLOCK * WARP1] = A[txyiA + WARP * (c + r * incA) + l * WARPS_PER_BLOCK * incA];
        }
        //load B(r)
        #pragma unroll
        for(l = 0; l < B_COLS_PER_WARP; l++)
          if(active_col > l)
            rB[l] = BB[txyiB + WARP * r + l * WARPS_PER_BLOCK * incB];
        __syncthreads();

        //gemm A(r,c)|A(c,r) & B(r) onto B(c) held at s
        if(active_col > 0){
          if(TRANS)
            sAA = sA + tx*WARP1;
          else
            sAA = sA + tx;
          #pragma unroll
          for(j = 0; j < WARP; j+=4){
            if(TRANS){
              #pragma unroll
              for(i = 0; i < 4; i++)
                //a[i] = CONJG ? conjugate(sAA[(j + i) * WARP1]) : sAA[(j + i) * WARP1];
                a[i] = CONJG ? conjugate(sAA[j + i]) : sAA[j + i];
            }
            else{
              #pragma unroll
              for(i = 0; i < 4; i++)
                a[i] = sAA[(j + i) * WARP1];
            }

            #pragma unroll
            for(l = 0; l < B_COLS_PER_WARP; l++){
                #pragma unroll
                for(i = 0; i < 4; i++)
                  b[i] = shfl(rB[l], j + i);
                #pragma unroll
                for(i = 0; i < 4; i++)
                  s[l] = FMA( a[i], b[i], s[l] );
            }
          }
        }
        __syncthreads();
      }
      //store back B(c) to global mem
      #pragma unroll
      for(l = 0; l < B_COLS_PER_WARP; l++){
        if(active_col > l){
          BB[txyiB + WARP * c + l * WARPS_PER_BLOCK * incB] = alpha * s[l];
        }
      }
    }
  }
}
//==============================================================================================
template<typename T, int WARPS_PER_BLOCK, int B_ROWS_PER_WARP, bool LOWER, bool TRANS, bool CONJG>
__global__ void //__launch_bounds__(256)
trmm_mul32_R(int M, int N, T alpha, const T* __restrict__ A, int incA, T* B, int incB, int nb){

  const int A_COL_PER_WARP = WARP / WARPS_PER_BLOCK;
  const int B_ROWS_PER_BLOCK = WARPS_PER_BLOCK * B_ROWS_PER_WARP;
  const bool forward = (LOWER != TRANS);
  
  int txyw = tx + ty*WARP1, tyxw = ty + tx*WARP1, txyiA = tx + ty*incA;
  //int txyiB = tx % B_ROWS_PER_BLOCK + (tx / B_ROWS_PER_BLOCK + ty * WARP / B_ROWS_PER_BLOCK) * incB;
  
  //setup shared memory
  __shared__ T sA[WARP * WARP1];//strided to avoid bank conflict
  T rB[B_ROWS_PER_WARP], rBj[B_ROWS_PER_WARP], s[B_ROWS_PER_WARP], a[4], b[4], *sAA, *BB;
  int c, j, r, l, i, startB = 0, active_row = 0;
  
  for(startB = 0; startB < M; startB += gridDim.x * B_ROWS_PER_BLOCK)
  {
    
    if( (startB + blockIdx.x * B_ROWS_PER_BLOCK) >= M) return;
    
    BB = B + (startB + blockIdx.x * B_ROWS_PER_BLOCK);
    active_row = 0;//an inactive warp will still contribute to data fetching but not to computation
    
    #pragma unroll
    for(l = 0; l < B_ROWS_PER_WARP; l++)
      active_row += ( (startB + blockIdx.x * B_ROWS_PER_BLOCK + ty + l * WARPS_PER_BLOCK) < M);

    for( c = (forward ? 0 : nb-1); (forward && (c < nb)) || (!forward && (c > -1)); c += (forward ? 1 : -1))
    {
      //load B(c) into registers in steps: 1. read coalesced into shared memory. 2. read into registers
      if( (blockIdx.x * B_ROWS_PER_BLOCK + tx % B_ROWS_PER_BLOCK) < M){
        #pragma unroll
        for(l = 0; l < B_ROWS_PER_WARP; l++)
          sA[tx % B_ROWS_PER_BLOCK + (tx / B_ROWS_PER_BLOCK + (ty + l * WARPS_PER_BLOCK) * WARP / B_ROWS_PER_BLOCK) * WARP1] = BB[tx % B_ROWS_PER_BLOCK + (tx / B_ROWS_PER_BLOCK + (ty + l * WARPS_PER_BLOCK) * WARP / B_ROWS_PER_BLOCK) * incB + WARP * c * incB];
      }
      __syncthreads();
      #pragma unroll
      for(l = 0; l < B_ROWS_PER_WARP; l++)
        rB[l] = sA[tyxw + l * WARPS_PER_BLOCK];
      __syncthreads();
      
      #pragma unroll
      for(l = 0; l < B_ROWS_PER_WARP; l++)
        s[l] = make_zero<T>();
      //load A(c,c) from global to shared mem
      #pragma unroll
      for(l = 0; l < A_COL_PER_WARP; l++)
        sA[txyw + l * WARPS_PER_BLOCK * WARP1] = A[txyiA + WARP * c * (incA+1) + l * WARPS_PER_BLOCK * incA];
      __syncthreads();
      if(!TRANS){
        #pragma unroll
        for(l = 0; l < A_COL_PER_WARP; l++)
          if(tx < (ty + l * WARPS_PER_BLOCK))
            sA[(ty + l * WARPS_PER_BLOCK) + WARP1 * tx] = sA[tx + WARP1 * (ty + l * WARPS_PER_BLOCK)];
      }
      __syncthreads();

      //perform trmm on shared mem
      if(active_row > 0){
        for(j = (forward ? 0 : WARP - 1); (forward && (j < WARP)) || (!forward && (j > -1)); j+=(forward ? 1 : -1)){
          #pragma unroll
          for(l = 0; l < B_ROWS_PER_WARP; l++)
            rBj[l] = shfl(rB[l], j);
          if( (forward && (j >= tx)) || (!forward && (j <= tx)) ){
            a[0] = CONJG ? conjugate(sA[tx + j * WARP1]) : sA[tx + j * WARP1];
            #pragma unroll
            for(l = 0; l < B_ROWS_PER_WARP; l++)
              s[l] = FMA( a[0], rBj[l], s[l]);
          }
        }
      }
      __syncthreads();

      for(r = (forward ? c+1 : 0); (forward && (r < nb)) || (!forward && (r < c)); r++){
        
        //load B(r) into registers in 2 steps: 1. read coalesced into shared memory. 2. read into registers
        if( (blockIdx.x * B_ROWS_PER_BLOCK + tx % B_ROWS_PER_BLOCK) < M){
          #pragma unroll
          for(l = 0; l < B_ROWS_PER_WARP; l++)
            sA[tx % B_ROWS_PER_BLOCK + (tx / B_ROWS_PER_BLOCK + (ty + l * WARPS_PER_BLOCK) * WARP / B_ROWS_PER_BLOCK) * WARP1] = BB[tx % B_ROWS_PER_BLOCK + (tx / B_ROWS_PER_BLOCK + (ty + l * WARPS_PER_BLOCK) * WARP / B_ROWS_PER_BLOCK) * incB + WARP * r * incB];
        }
        __syncthreads();
        #pragma unroll
        for(l = 0; l < B_ROWS_PER_WARP; l++)
          rB[l] = sA[tyxw + l * WARPS_PER_BLOCK];
        __syncthreads();
        
        #pragma unroll
        for(l = 0; l < A_COL_PER_WARP; l++){
          if(!TRANS)//load A(r,c)
            //sA[tyxw + l * WARPS_PER_BLOCK] = A[txyiA + WARP * (r + c * incA) + l * WARPS_PER_BLOCK * incA];
            sA[txyw + l * WARPS_PER_BLOCK * WARP1] = A[txyiA + WARP * (r + c * incA) + l * WARPS_PER_BLOCK * incA];
          else//load A(c,r)
            sA[txyw + l * WARPS_PER_BLOCK * WARP1] = A[txyiA + WARP * (c + r * incA) + l * WARPS_PER_BLOCK * incA];
        }
        __syncthreads();

        //gemm B(r) & A(r,c)|A(c,r) onto B(c) held at s
        if(active_row > 0){
          if(!TRANS)
            sAA = sA + tx*WARP1;
          else
            sAA = sA + tx;
          #pragma unroll
          for(j = 0; j < WARP; j+=4){
            if(!TRANS){
              #pragma unroll
              for(i = 0; i < 4; i++)
                //a[i] = CONJG ? conjugate(sAA[(j + i) * WARP1]) : sAA[(j + i) * WARP1];
                a[i] = CONJG ? conjugate(sAA[j + i]) : sAA[j + i];
            }
            else{
              #pragma unroll
              for(i = 0; i < 4; i++)
                a[i] = sAA[(j + i) * WARP1];
            }

            #pragma unroll
            for(l = 0; l < B_ROWS_PER_WARP; l++){
                #pragma unroll
                for(i = 0; i < 4; i++)
                  b[i] = shfl(rB[l], j + i);
                #pragma unroll
                for(i = 0; i < 4; i++)
                  s[l] = FMA( a[i], b[i], s[l] );
            }
          }
        }
        __syncthreads();
      }
      
      //store back B(c) to global mem in 2 steps: 1. store in shared memory 2. read from shared memory to global memory
      #pragma unroll
      for(l = 0; l < B_ROWS_PER_WARP; l++)
        sA[tyxw + l * WARPS_PER_BLOCK] = alpha * s[l];
      __syncthreads();
      if( (blockIdx.x * B_ROWS_PER_BLOCK + tx % B_ROWS_PER_BLOCK) < M){
        #pragma unroll
        for(l = 0; l < B_ROWS_PER_WARP; l++)
          BB[tx % B_ROWS_PER_BLOCK + (tx / B_ROWS_PER_BLOCK + (ty + l * WARPS_PER_BLOCK) * WARP / B_ROWS_PER_BLOCK) * incB + WARP * c * incB] = sA[tx % B_ROWS_PER_BLOCK + (tx / B_ROWS_PER_BLOCK + (ty + l * WARPS_PER_BLOCK) * WARP / B_ROWS_PER_BLOCK) * WARP1];
      }
      __syncthreads();
    }
  }
}
//==============================================================================================
template<class T>
hipblasStatus_t Xtrmm(hipblasHandle_t handle,
                     hipblasSideMode_t side, hipblasFillMode_t uplo,
                     hipblasOperation_t trans, hipblasDiagType_t diag,
                     int m, int n,
                     const T *alpha, const T *A, int incA,
                                           T *B, int incB)
{
  //handle odd cases with cublas
  if(  (*alpha == make_zero<T>())
    || (!kblas_trmm_use_custom)
    || (side == HIPBLAS_SIDE_LEFT && m < WARP)
    || (side == HIPBLAS_SIDE_RIGHT && n < WARP)){
    return cublasXtrmm(handle,
                       side, uplo, trans, diag,
                       m, n,
                       alpha, A, incA,
                              B, incB );
  }
  
  typedef void (*trmm_kernels_type)(int M, int N, T alpha, const T* A, int incA, T* B, int incB, int mb);

  #define WARPS_PER_BLOCK 8
  #define B_COLS_PER_WARP 1
  
  trmm_kernels_type trmm_kernels[8] = {// T, WARPS_PER_BLOCK, B_COLS_PER_WARP, LEFT, LOWER, TRANS, CONJG
    trmm_mul32_L<T, WARPS_PER_BLOCK, B_COLS_PER_WARP,  true, false, false>,
    trmm_mul32_L<T, WARPS_PER_BLOCK, B_COLS_PER_WARP,  true,  true, false>,
    trmm_mul32_L<T, WARPS_PER_BLOCK, B_COLS_PER_WARP, false, false, false>,
    trmm_mul32_L<T, WARPS_PER_BLOCK, B_COLS_PER_WARP, false,  true, false>,
    trmm_mul32_R<T, WARPS_PER_BLOCK, B_COLS_PER_WARP,  true, false, false>,
    trmm_mul32_R<T, WARPS_PER_BLOCK, B_COLS_PER_WARP,  true,  true, false>,
    trmm_mul32_R<T, WARPS_PER_BLOCK, B_COLS_PER_WARP, false, false, false>,
    trmm_mul32_R<T, WARPS_PER_BLOCK, B_COLS_PER_WARP, false,  true, false>
  };
  
  hipStream_t curStream;
  hipblasStatus_t status;

  check_error( status = hipblasGetStream( handle, &curStream ), status);
  
  if( ((side == HIPBLAS_SIDE_LEFT) && (m % WARP == 0)) || ((side == HIPBLAS_SIDE_RIGHT) && (n % WARP == 0)))
  {
    int func_idx = 4*(side == HIPBLAS_SIDE_RIGHT) + 2*(uplo == HIPBLAS_FILL_MODE_UPPER) + (trans != HIPBLAS_OP_N);// + (diag == HIPBLAS_DIAG_UNIT);
    dim3 blockDim( WARP, WARPS_PER_BLOCK );
    dim3 gridDim(
      (side == HIPBLAS_SIDE_LEFT) * (n / (WARPS_PER_BLOCK * B_COLS_PER_WARP) + (n % (WARPS_PER_BLOCK * B_COLS_PER_WARP) > 0))
      +
      (side == HIPBLAS_SIDE_RIGHT) * (m / (WARPS_PER_BLOCK * B_COLS_PER_WARP) + (m % (WARPS_PER_BLOCK * B_COLS_PER_WARP) > 0))
      , 1);
    int mb = (side == HIPBLAS_SIDE_LEFT) * m / WARP + (side == HIPBLAS_SIDE_RIGHT) * n / WARP;
    //TODO validate with this run from magma ./testing/testing_dpotri_gpu --dev 1 --range 512:15360:512
    trmm_kernels[func_idx]<<< gridDim, blockDim, 0, curStream>>> (m, n, *alpha, A, incA, B, incB, mb);
    check_error( hipGetLastError(),  HIPBLAS_STATUS_EXECUTION_FAILED );
  }else{
    //error: we should not reach this case
    return HIPBLAS_STATUS_INTERNAL_ERROR;
  }
  return HIPBLAS_STATUS_SUCCESS;
}
#else
template<class T>
hipblasStatus_t Xtrmm(hipblasHandle_t handle,
                     hipblasSideMode_t side, hipblasFillMode_t uplo,
                     hipblasOperation_t trans, hipblasDiagType_t diag,
                     int m, int n,
                     const T *alpha, const T *A, int incA,
                     T *B, int incB)
{
  return cublasXtrmm( handle,
                      side, uplo, trans, diag,
                      m, n,
                      alpha, A, incA,
                             B, incB );
}

#endif //(SM >= 30)

//==============================================================================================
template<class T>
hipblasStatus_t kblasXtrmm(hipblasHandle_t handle,
                          hipblasSideMode_t side, hipblasFillMode_t uplo,
                          hipblasOperation_t trans, hipblasDiagType_t diag,
                          int m, int n,
                          const T *alpha,
                          const T *A, int incA,
                                T *B, int incB)
{
  T one = make_one<T>();
  hipblasStatus_t status;
  
  if( (*alpha == make_zero<T>())//TODO
   || ( (side == HIPBLAS_SIDE_LEFT) && (SIMPLE_SIZE(m)) )
   || ( (side == HIPBLAS_SIDE_RIGHT) && (SIMPLE_SIZE(n)) ) ){
    return Xtrmm(handle,
                 side, uplo, trans, diag,
                 m, n,
                 alpha, A, incA,
                        B, incB );
  }else
  if(side == HIPBLAS_SIDE_LEFT){

    int m1, m2;
    if(REG_SIZE(m))
      m1 = m2 = m/2;
    else{
      m1 = CLOSEST_REG_SIZE(m);
      m2 = m-m1;
    }
    hipblasOperation_t noTrans = HIPBLAS_OP_N;//Trans = HIPBLAS_OP_T,

    if(uplo == HIPBLAS_FILL_MODE_UPPER){

      //Left / Upper / NoTrans
      if(trans == HIPBLAS_OP_N){
        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m1, n,
                                alpha, A, incA,
                                       B, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = cublasXgemm(handle,
                                 trans, noTrans,
                                 m1, n, m2,
                                 alpha, A+m1*incA, incA,
                                        B+m1, incB,
                                 &one,  B, incB)) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m2, n,
                                alpha, A+m1+m1*incA, incA,
                                       B+m1, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
      //Left / Upper / [Conj]Trans
      else{
        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m2, n,
                                alpha, A+m1+m1*incA, incA,
                                       B+m1, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = cublasXgemm(handle,
                                 trans, noTrans,
                                 m2, n, m1,
                                 alpha, A+m1*incA, incA,
                                        B, incB,
                                 &one,  B+m1, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m1, n,
                                alpha, A, incA,
                                       B, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }

    }else{//uplo == Lower

      //Left / Lower / NoTrans
      if(trans == HIPBLAS_OP_N){
        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m2, n,
                                alpha, A+m1+m1*incA, incA,
                                       B+m1, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = cublasXgemm(handle,
                                 trans, noTrans,
                                 m2, n, m1,
                                 alpha, A+m1, incA,
                                        B, incB,
                                 &one,  B+m1, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m1, n,
                                alpha, A, incA,
                                       B, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
      //Left / Lower / [Conj]Trans
      else{//trans == Trans
        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m1, n,
                                alpha, A, incA,
                                       B, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = cublasXgemm(handle,
                                 trans, noTrans,
                                 m1, n, m2,
                                 alpha, A+m1, incA,
                                        B+m1, incB,
                                 &one,  B, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m2, n,
                                alpha, A+m1+m1*incA, incA,
                                       B+m1, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }//trans == Trans
    }//uplo == Lower

  }else{//side == Right
    int n1, n2;

    if(REG_SIZE(n))
      n1 = n2 = n/2;
    else{
      n1 = CLOSEST_REG_SIZE(n);
      n2 = n-n1;
    }

    if(uplo == HIPBLAS_FILL_MODE_UPPER){
      //Right / Upper / NoTrans
      if(trans == HIPBLAS_OP_N){
        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m, n2,
                                alpha, A+n1+n1*incA, incA,
                                       B+n1*incB, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = cublasXgemm(handle,
                                 HIPBLAS_OP_N, trans,
                                 m, n2, n1,
                                 alpha, B, incB,
                                        A+n1*incA, incA,
                                 &one,  B+n1*incB, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m, n1,
                                alpha, A, incA,
                                       B, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
      //Right / Upper / [Conj]Trans
      else{
        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m, n1,
                                alpha, A, incA,
                                       B, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = cublasXgemm(handle,
                                 HIPBLAS_OP_N, trans,
                                 m, n1, n2,
                                 alpha, B+n1*incB, incB,
                                        A+n1*incA, incA,
                                 &one,  B, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m, n2,
                                alpha, A+n1+n1*incA, incA,
                                       B+n1*incB, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
    }else{
      //Right / Lower / NoTrans
      if(trans == HIPBLAS_OP_N){
        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m, n1,
                                alpha, A, incA,
                                       B, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = cublasXgemm(handle,
                                 HIPBLAS_OP_N, trans,
                                 m, n1, n2,
                                 alpha, B+n1*incB, incB,
                                        A+n1, incA,
                                 &one,  B, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m, n2,
                                alpha, A+n1+n1*incA, incA,
                                       B+n1*incB, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
      //Right / Lower / [Conj]Trans
      else{
        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m, n2,
                                alpha, A+n1+n1*incA, incA,
                                       B+n1*incB, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = cublasXgemm(handle,
                                 HIPBLAS_OP_N, trans,
                                 m, n2, n1,
                                 alpha, B, incB,
                                        A+n1, incA,
                                 &one,  B+n1*incB, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m, n1,
                                alpha, A, incA,
                                       B, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
    }

  }//side == Right

  return HIPBLAS_STATUS_SUCCESS;
}

//==============================================================================================
template<class T>
hipblasStatus_t kblasXtrmm(hipblasHandle_t handle, hipStream_t &strIn, hipStream_t &strOut,
                          hipblasSideMode_t side, hipblasFillMode_t uplo,
                          hipblasOperation_t trans, hipblasDiagType_t diag,
                          int m, int n,
                          const T *alpha,
                          const T *h_A, int incA, T* d_A, 
                                T *h_B, int incB, T* d_B,
                          bool BIsIn, bool getBOut, bool AIsIn)
{
  T one = make_one<T>();
  hipblasStatus_t status;
  hipEvent_t eDataIn, eComp;
  check_error( hipEventCreateWithFlags(&eDataIn, hipEventDisableTiming), HIPBLAS_STATUS_EXECUTION_FAILED);
  check_error( hipEventCreateWithFlags(&eComp, hipEventDisableTiming), HIPBLAS_STATUS_EXECUTION_FAILED);
  hipStream_t strComp;
  check_error( hipblasGetStream(handle, &strComp), HIPBLAS_STATUS_INTERNAL_ERROR);
  
  if( ( *alpha == make_zero<T>() ) //TODO
   || ( (side == HIPBLAS_SIDE_LEFT) && (SIMPLE_SIZE(m)) ) 
   || ( (side == HIPBLAS_SIDE_RIGHT) && (SIMPLE_SIZE(n)) ) ){

    int Am = (side == HIPBLAS_SIDE_LEFT) ? m : n;
    //if B is not already in, copy in B block
    if(!BIsIn)
      check_error( status = hipblasSetMatrixAsync( m, n, sizeof(T), h_B, incB, d_B, incB, strIn ), status);
    //copy in A block
    if(!AIsIn)
      check_error( status = hipblasSetMatrixAsync( Am, Am, sizeof(T), h_A, incA, d_A, incA, strIn ), status);
    //wait for data to arrive
    if(!AIsIn || !BIsIn){
      check_error( hipEventRecord(eDataIn, strIn), HIPBLAS_STATUS_INTERNAL_ERROR);
      check_error( hipStreamWaitEvent(strComp, eDataIn, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
    }
    if( (status = Xtrmm( handle,
                         side, uplo, trans, diag,
                         m, n,
                         alpha, d_A, incA,
                                d_B, incB ) ) != HIPBLAS_STATUS_SUCCESS ) return status;

    //if stream is done computing and getBOut, copy B back.
    if(getBOut){
      check_error( hipEventRecord(eComp, strComp), HIPBLAS_STATUS_INTERNAL_ERROR);
      check_error( hipStreamWaitEvent(strOut, eComp, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
      check_error( status = hipblasGetMatrixAsync( m, n, sizeof(T), d_B, incB, h_B, incB, strOut), status);
    }
  }else
  if(side == HIPBLAS_SIDE_LEFT){

    int m1, m2;
    if(REG_SIZE(m))
      m1 = m2 = m/2;
    else{
      m1 = CLOSEST_REG_SIZE(m);
      m2 = m-m1;
    }
    hipblasOperation_t noTrans = HIPBLAS_OP_N;//Trans = HIPBLAS_OP_T,

    if( (!AIsIn && SIMPLE_SIZE_DATA(m)) || (!BIsIn && SIMPLE_SIZE_DATA(m)) ){
      if( (!AIsIn && SIMPLE_SIZE_DATA(m)) ){
        check_error( status = hipblasSetMatrixAsync( m, m, sizeof(T), h_A, incA, d_A, incA, strIn), status);
        AIsIn = true;
      }
      if( (!BIsIn && SIMPLE_SIZE_DATA(m)) ){
        check_error( status = hipblasSetMatrixAsync( m, n, sizeof(T), h_B, incB, d_B, incB, strIn), status);
        BIsIn = true;
      }
      //wait for data to arrive
      check_error( hipEventRecord(eDataIn, strIn), HIPBLAS_STATUS_INTERNAL_ERROR);
      check_error( hipStreamWaitEvent(strComp, eDataIn, 0), HIPBLAS_STATUS_INTERNAL_ERROR);      
    }
    if(uplo == HIPBLAS_FILL_MODE_UPPER){

      //Left / Upper / NoTrans
      if(trans == HIPBLAS_OP_N){
        if((status = kblasXtrmm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m1, n,
                                alpha, h_A, incA, d_A,
                                       h_B, incB, d_B,
                                BIsIn, false, AIsIn
                                )) != HIPBLAS_STATUS_SUCCESS) return status;

        //prepare needed data
        if(!AIsIn || !BIsIn){
          //if B is not already in, copy B block
          if(!BIsIn){
            check_error( status = hipblasSetMatrixAsync( m2, n, sizeof(T), h_B + m1, incB, d_B + m1, incB, strIn), status);
            BIsIn = true;
          }
          //copy in A block
          if(!AIsIn)
            check_error( status = hipblasSetMatrixAsync( m1, m2, sizeof(T), h_A + m1 * incA, incA, d_A + m1 * incA, incA, strIn), status);
          //wait for data to arrive
          check_error( hipEventRecord(eDataIn, strIn), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strComp, eDataIn, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
        }
        if((status = cublasXgemm(handle,
                                 trans, noTrans,
                                 m1, n, m2,
                                 alpha, d_A + m1 * incA, incA,
                                        d_B + m1, incB,
                                 &one,  d_B, incB)) != HIPBLAS_STATUS_SUCCESS) return status;
        //if stream is done computing and getBOut, copy B back.
        if(getBOut){
          check_error( hipEventRecord(eComp, strComp), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strOut, eComp, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( status = hipblasGetMatrixAsync( m1, n, sizeof(T), d_B, incB, h_B, incB, strOut), status);
        }

        //B is already in, no need to copy in
        if((status = kblasXtrmm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m2, n,
                                alpha, h_A + m1 + m1 * incA, incA, d_A + m1 + m1 * incA,
                                       h_B + m1, incB, d_B + m1,
                                BIsIn, getBOut, AIsIn
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
      //Left / Upper / [Conj]Trans
      else{
        if((status = kblasXtrmm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m2, n,
                                alpha, h_A+m1+m1*incA, incA, d_A+m1+m1*incA,
                                       h_B+m1, incB, d_B+m1,
                                BIsIn, false, AIsIn
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
        //prepare needed data
        if(!AIsIn || !BIsIn){
          //if B is not already in, copy B block
          if(!BIsIn){
            check_error( status = hipblasSetMatrixAsync( m1, n, sizeof(T), h_B, incB, d_B, incB, strIn), status);
            BIsIn = true;
          }
          //copy in A block
          if(!AIsIn)
            check_error( status = hipblasSetMatrixAsync( m1, m2, sizeof(T), h_A + m1 * incA, incA, d_A + m1 * incA, incA, strIn), status);
          //wait for data to arrive
          check_error( hipEventRecord(eDataIn, strIn), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strComp, eDataIn, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
        }
        if((status = cublasXgemm(handle,
                                 trans, noTrans,
                                 m2, n, m1,
                                 alpha, d_A+m1*incA, incA,
                                        d_B, incB,
                                 &one,  d_B+m1, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        //if stream is done computing and getBOut, copy B back.
        if(getBOut){
          check_error( hipEventRecord(eComp, strComp), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strOut, eComp, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( status = hipblasGetMatrixAsync( m2, n, sizeof(T), d_B+m1, incB, h_B+m1, incB, strOut), status);
        }

        if((status = kblasXtrmm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m1, n,
                                alpha, h_A, incA, d_A,
                                       h_B, incB, d_B,
                                BIsIn, getBOut, AIsIn
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }

    }else{//uplo == Lower

      //Left / Lower / NoTrans
      if(trans == HIPBLAS_OP_N){
        if((status = kblasXtrmm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m2, n,
                                alpha, h_A+m1+m1*incA, incA, d_A+m1+m1*incA, 
                                       h_B+m1, incB, d_B+m1,
                                BIsIn, false, AIsIn
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
        //prepare needed data
        if(!AIsIn || !BIsIn){
          //if B is not already in, copy B block
          if(!BIsIn){
            check_error( status = hipblasSetMatrixAsync( m1, n, sizeof(T), h_B, incB, d_B, incB, strIn), status);
            BIsIn = true;
          }
          //copy in A block
          if(!AIsIn)
            check_error( status = hipblasSetMatrixAsync( m2, m1, sizeof(T), h_A + m1, incA, d_A + m1, incA, strIn), status);
          //wait for data to arrive
          check_error( hipEventRecord(eDataIn, strIn), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strComp, eDataIn, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
        }
        if((status = cublasXgemm(handle,
                                 trans, noTrans,
                                 m2, n, m1,
                                 alpha, d_A+m1, incA,
                                        d_B, incB,
                                 &one,  d_B+m1, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;
        //if stream is done computing and getBOut, copy B back.
        if(getBOut){
          check_error( hipEventRecord(eComp, strComp), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strOut, eComp, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( status = hipblasGetMatrixAsync( m2, n, sizeof(T), d_B+m1, incB, h_B+m1, incB, strOut), status);
        }

        if((status = kblasXtrmm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m1, n,
                                alpha, h_A, incA, d_A,
                                       h_B, incB, d_B,
                                BIsIn, getBOut, AIsIn
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
      //Left / Lower / [Conj]Trans
      else{//trans == Trans
        if((status = kblasXtrmm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m1, n,
                                alpha, h_A, incA, d_A,
                                       h_B, incB, d_B,
                                BIsIn, false, AIsIn
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
        //prepare needed data
        if(!AIsIn || !BIsIn){
          //if B is not already in, copy B block
          if(!BIsIn){
            check_error( status = hipblasSetMatrixAsync( m2, n, sizeof(T), h_B + m1, incB, d_B + m1, incB, strIn), status);
            BIsIn = true;
          }
          //copy in A block
          if(!AIsIn)
            check_error( status = hipblasSetMatrixAsync( m2, m1, sizeof(T), h_A + m1, incA, d_A + m1, incA, strIn), status);
          //wait for data to arrive
          check_error( hipEventRecord(eDataIn, strIn), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strComp, eDataIn, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
        }

        if((status = cublasXgemm(handle,
                                 trans, noTrans,
                                 m1, n, m2,
                                 alpha, d_A+m1, incA,
                                        d_B+m1, incB,
                                 &one,  d_B, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;
        //if stream is done computing and getBOut, copy B back.
        if(getBOut){
          check_error( hipEventRecord(eComp, strComp), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strOut, eComp, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( status = hipblasGetMatrixAsync( m1, n, sizeof(T), d_B, incB, h_B, incB, strOut), status);
        }

        if((status = kblasXtrmm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m2, n,
                                alpha, h_A+m1+m1*incA, incA, d_A+m1+m1*incA, 
                                       h_B+m1, incB, d_B+m1,
                                BIsIn, getBOut, AIsIn
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }//trans == Trans
    }//uplo == Lower

  }else{//side == Right
    int n1, n2;

    if(REG_SIZE(n))
      n1 = n2 = n/2;
    else{
      n1 = CLOSEST_REG_SIZE(n);
      n2 = n-n1;
    }
    
    if( (!AIsIn && SIMPLE_SIZE_DATA(n)) || (!BIsIn && SIMPLE_SIZE_DATA(n)) ){
      if( (!AIsIn && SIMPLE_SIZE_DATA(n)) ){
        check_error( status = hipblasSetMatrixAsync( n, n, sizeof(T), h_A, incA, d_A, incA, strIn), status);
        AIsIn = true;
      }
      if( (!BIsIn && SIMPLE_SIZE_DATA(n)) ){
        check_error( status = hipblasSetMatrixAsync( m, n, sizeof(T), h_B, incB, d_B, incB, strIn), status);
        BIsIn = true;
      }
      //wait for data to arrive
      check_error( hipEventRecord(eDataIn, strIn), HIPBLAS_STATUS_INTERNAL_ERROR);
      check_error( hipStreamWaitEvent(strComp, eDataIn, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
    }
    if(uplo == HIPBLAS_FILL_MODE_UPPER){
      //Right / Upper / NoTrans
      if(trans == HIPBLAS_OP_N){
        if((status = kblasXtrmm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m, n2,
                                alpha, h_A+n1+n1*incA, incA, d_A+n1+n1*incA, 
                                       h_B+n1*incB, incB, d_B+n1*incB,
                                BIsIn, false, AIsIn
                                )) != HIPBLAS_STATUS_SUCCESS) return status;

        //prepare needed data
        if(!AIsIn || !BIsIn){
          //if B is not already in, copy B block
          if(!BIsIn){
            check_error( status = hipblasSetMatrixAsync( m, n1, sizeof(T), h_B, incB, d_B, incB, strIn), status);
            BIsIn = true;
          }
          //copy in A block
          if(!AIsIn)
            check_error( status = hipblasSetMatrixAsync( n1, n2, sizeof(T), h_A + n1 * incA, incA, d_A + n1 * incA, incA, strIn), status);
          //wait for data to arrive
          check_error( hipEventRecord(eDataIn, strIn), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strComp, eDataIn, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
        }
        if((status = cublasXgemm(handle,
                                 HIPBLAS_OP_N, trans,
                                 m, n2, n1,
                                 alpha, d_B, incB,
                                        d_A+n1*incA, incA,
                                 &one,  d_B+n1*incB, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;
        //if stream is done computing and getBOut, copy B back.
        if(getBOut){
          check_error( hipEventRecord(eComp, strComp), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strOut, eComp, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( status = hipblasGetMatrixAsync( m, n2, sizeof(T), d_B+n1*incB, incB, h_B+n1*incB, incB, strOut), status);
        }

        if((status = kblasXtrmm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m, n1,
                                alpha, h_A, incA, d_A,
                                       h_B, incB, d_B,
                                BIsIn, getBOut, AIsIn
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
      //Right / Upper / [Conj]Trans
      else{
        if((status = kblasXtrmm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m, n1,
                                alpha, h_A, incA, d_A,
                                       h_B, incB, d_B,
                                BIsIn, false, AIsIn
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
        //prepare needed data
        if(!AIsIn || !BIsIn){
          //if B is not already in, copy B block
          if(!BIsIn){
            check_error( status = hipblasSetMatrixAsync( m, n2, sizeof(T), h_B+n1*incB, incB, d_B+n1*incB, incB, strIn), status);
            BIsIn = true;
          }
          //copy in A block
          if(!AIsIn)
            check_error( status = hipblasSetMatrixAsync( n1, n2, sizeof(T), h_A+n1*incA, incA, d_A+n1*incA, incA, strIn), status);
          //wait for data to arrive
          check_error( hipEventRecord(eDataIn, strIn), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strComp, eDataIn, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
        }
        if((status = cublasXgemm(handle,
                                 HIPBLAS_OP_N, trans,
                                 m, n1, n2,
                                 alpha, d_B+n1*incB, incB,
                                        d_A+n1*incA, incA,
                                 &one,  d_B, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;
        //if stream is done computing and getBOut, copy B back.
        if(getBOut){
          check_error( hipEventRecord(eComp, strComp), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strOut, eComp, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( status = hipblasGetMatrixAsync( m, n1, sizeof(T), d_B, incB, h_B, incB, strOut), status);
        }

        if((status = kblasXtrmm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m, n2,
                                alpha, h_A+n1+n1*incA, incA, d_A+n1+n1*incA, 
                                       h_B+n1*incB, incB, d_B+n1*incB,
                                BIsIn, getBOut, AIsIn
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
    }else{
      //Right / Lower / NoTrans
      if(trans == HIPBLAS_OP_N){
        if((status = kblasXtrmm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m, n1,
                                alpha, h_A, incA, d_A, 
                                       h_B, incB, d_B,
                                BIsIn, false, AIsIn
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
        //prepare needed data
        if(!AIsIn || !BIsIn){
          //if B is not already in, copy B block
          if(!BIsIn){
            check_error( status = hipblasSetMatrixAsync( m, n2, sizeof(T), h_B+n1*incB, incB, d_B+n1*incB, incB, strIn), status);
            BIsIn = true;
          }
          //copy in A block
          if(!AIsIn)
            check_error( status = hipblasSetMatrixAsync( n2, n1, sizeof(T), h_A+n1, incA, d_A+n1, incA, strIn), status);
          //wait for data to arrive
          check_error( hipEventRecord(eDataIn, strIn), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strComp, eDataIn, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
        }
        if((status = cublasXgemm(handle,
                                 HIPBLAS_OP_N, trans,
                                 m, n1, n2,
                                 alpha, d_B+n1*incB, incB,
                                        d_A+n1, incA,
                                 &one,  d_B, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;
        //if stream is done computing and getBOut, copy B back.
        if(getBOut){
          check_error( hipEventRecord(eComp, strComp), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strOut, eComp, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( status = hipblasGetMatrixAsync( m, n1, sizeof(T), d_B, incB, h_B, incB, strOut), status);
        }

        if((status = kblasXtrmm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m, n2,
                                alpha, h_A+n1+n1*incA, incA, d_A+n1+n1*incA, 
                                       h_B+n1*incB, incB, d_B+n1*incB,
                                BIsIn, getBOut, AIsIn
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
      //Right / Lower / [Conj]Trans
      else{
        if((status = kblasXtrmm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m, n2,
                                alpha, h_A+n1+n1*incA, incA, d_A+n1+n1*incA, 
                                       h_B+n1*incB, incB, d_B+n1*incB,
                                BIsIn, false, AIsIn
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
        //prepare needed data
        if(!AIsIn || !BIsIn){
          //if B is not already in, copy B block
          if(!BIsIn){
            check_error( status = hipblasSetMatrixAsync( m, n1, sizeof(T), h_B, incB, d_B, incB, strIn), status);
            BIsIn = true;
          }
          //copy in A block
          if(!AIsIn)
            check_error( status = hipblasSetMatrixAsync( n1, n2, sizeof(T), h_A+n1, incA, d_A+n1, incA, strIn), status);
          //wait for data to arrive
          check_error( hipEventRecord(eDataIn, strIn), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strComp, eDataIn, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
        }
        if((status = cublasXgemm(handle,
                                 HIPBLAS_OP_N, trans,
                                 m, n2, n1,
                                 alpha, d_B, incB,
                                        d_A+n1, incA,
                                 &one,  d_B+n1*incB, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;
        //if stream is done computing and getBOut, copy B back.
        if(getBOut){
          check_error( hipEventRecord(eComp, strComp), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strOut, eComp, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( status = hipblasGetMatrixAsync( m, n2, sizeof(T), d_B+n1*incB, incB, h_B+n1*incB, incB, strOut), status);
        }

        if((status = kblasXtrmm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m, n1,
                                alpha, h_A, incA, d_A,
                                       h_B, incB, d_B,
                                BIsIn, getBOut, AIsIn
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
    }

  }//side == Right

  check_error( hipEventDestroy( eDataIn ), HIPBLAS_STATUS_INTERNAL_ERROR);
  check_error( hipEventDestroy( eComp ), HIPBLAS_STATUS_INTERNAL_ERROR);
  check_error( hipGetLastError(), HIPBLAS_STATUS_EXECUTION_FAILED );
  return HIPBLAS_STATUS_SUCCESS;
}


/*/==============================================================================================
template<class T>
hipblasStatus_t kblasXtrmm(hipblasHandle_t handle, hipStream_t &strIn, hipStream_t &strOut,
                          hipblasSideMode_t side, hipblasFillMode_t uplo,
                          hipblasOperation_t trans, hipblasDiagType_t diag,
                          int m, int n,
                          const T *alpha,
                          const T *Ac, int incA, T* Ad, 
                                T *Bc, int incB, T* Bd, bool Bin, bool Bout)//TODO wait on intial event
{
  T one = make_one<T>();
  hipblasStatus_t status;
  hipEvent_t eDataIn, eComp;
  hipStream_t inStream;
  check_error( hipStreamCreateWithFlags( &inStream, hipStreamNonBlocking), HIPBLAS_STATUS_EXECUTION_FAILED );
  check_error( hipEventCreateWithFlags(&eDataIn, hipEventDisableTiming), HIPBLAS_STATUS_EXECUTION_FAILED);
  check_error( hipEventCreateWithFlags(&eComp, hipEventDisableTiming), HIPBLAS_STATUS_EXECUTION_FAILED);
  hipStream_t compStream;
  check_error( hipblasGetStream(handle, &compStream), HIPBLAS_STATUS_INTERNAL_ERROR);
  
  if( ( *alpha == make_zero<T>() ) //TODO
   || ( (side == HIPBLAS_SIDE_LEFT) && (SIMPLE_SIZE(m)) ) 
   || ( (side == HIPBLAS_SIDE_RIGHT) && (SIMPLE_SIZE(n)) ) ){

    int Am = (side == HIPBLAS_SIDE_LEFT) ? m : n;
    //if B is not already in, copy in B block
    if(!Bin)
      check_error( status = hipblasSetMatrixAsync( m, n, sizeof(T), Bc, incB, Bd, incB, inStream ), status);
    //copy in A block
    check_error( status = hipblasSetMatrixAsync( Am, Am, sizeof(T), Ac, incA, Ad, incA, inStream ), status);
    //wait for data to arrive
    check_error( hipEventRecord(eDataIn, inStream), HIPBLAS_STATUS_INTERNAL_ERROR);
    check_error( hipStreamWaitEvent(compStream, eDataIn, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
  
    if( (status = Xtrmm( handle,
                         side, uplo, trans, diag,
                         m, n,
                         alpha, Ad, incA,
                                Bd, incB ) ) != HIPBLAS_STATUS_SUCCESS ) return status;

    //if stream is done computing and Bout, copy B back.
    if(Bout){
      check_error( hipEventRecord(eComp, compStream), HIPBLAS_STATUS_INTERNAL_ERROR);
      check_error( hipStreamWaitEvent(strOut, eComp, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
      check_error( status = hipblasGetMatrixAsync( m, n, sizeof(T), Bd, incB, Bc, incB, strOut), status);
    }
  }else
  if(side == HIPBLAS_SIDE_LEFT){

    int m1, m2;
    if(REG_SIZE(m))
      m1 = m2 = m/2;
    else{
      m1 = CLOSEST_REG_SIZE(m);
      m2 = m-m1;
    }
    hipblasOperation_t noTrans = HIPBLAS_OP_N;//Trans = HIPBLAS_OP_T,

    if(uplo == HIPBLAS_FILL_MODE_UPPER){

      //Left / Upper / NoTrans
      if(trans == HIPBLAS_OP_N){
        if((status = kblasXtrmm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m1, n,
                                alpha, Ac, incA, Ad,
                                       Bc, incB, Bd, false, false
                                )) != HIPBLAS_STATUS_SUCCESS) return status;

        //if B is not already in, copy B block
        if(!Bin)
          check_error( status = hipblasSetMatrixAsync( m2, n, sizeof(T), Bc + m1, incB, Bd + m1, incB, inStream), status);
        //copy in A block
        check_error( status = hipblasSetMatrixAsync( m1, m2, sizeof(T), Ac + m1 * incA, incA, Ad + m1 * incA, incA, inStream), status);
        //wait for data to arrive
        check_error( hipEventRecord(eDataIn, inStream), HIPBLAS_STATUS_INTERNAL_ERROR);
        check_error( hipStreamWaitEvent(compStream, eDataIn, 0), HIPBLAS_STATUS_INTERNAL_ERROR);

        if((status = cublasXgemm(handle,
                                 trans, noTrans,
                                 m1, n, m2,
                                 alpha, Ad + m1 * incA, incA,
                                        Bd + m1, incB,
                                 &one,  Bd, incB)) != HIPBLAS_STATUS_SUCCESS) return status;
        //if stream is done computing and Bout, copy B back.
        if(Bout){
          check_error( hipEventRecord(eComp, compStream), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strOut, eComp, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( status = hipblasGetMatrixAsync( m1, n, sizeof(T), Bd, incB, Bc, incB, strOut), status);
        }

        //B is already in, no need to copy in
        if((status = kblasXtrmm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m2, n,
                                alpha, Ac + m1 + m1 * incA, incA, Ad + m1 + m1 * incA,
                                       Bc + m1, incB, Bd + m1, true, Bout
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
      //Left / Upper / [Conj]Trans
      else{
        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m2, n,
                                alpha, Ac+m1+m1*incA, incA,
                                       Bc+m1, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = cublasXgemm(handle,
                                 trans, noTrans,
                                 m2, n, m1,
                                 alpha, Ac+m1*incA, incA,
                                        Bc, incB,
                                 &one,  Bc+m1, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m1, n,
                                alpha, Ac, incA,
                                       Bc, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }

    }else{//uplo == Lower

      //Left / Lower / NoTrans
      if(trans == HIPBLAS_OP_N){
        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m2, n,
                                alpha, Ac+m1+m1*incA, incA,
                                       Bc+m1, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = cublasXgemm(handle,
                                 trans, noTrans,
                                 m2, n, m1,
                                 alpha, Ac+m1, incA,
                                        Bc, incB,
                                 &one,  Bc+m1, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m1, n,
                                alpha, Ac, incA,
                                       Bc, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
      //Left / Lower / [Conj]Trans
      else{//trans == Trans
        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m1, n,
                                alpha, Ac, incA,
                                       Bc, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = cublasXgemm(handle,
                                 trans, noTrans,
                                 m1, n, m2,
                                 alpha, Ac+m1, incA,
                                        Bc+m1, incB,
                                 &one,  Bc, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m2, n,
                                alpha, Ac+m1+m1*incA, incA,
                                       Bc+m1, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }//trans == Trans
    }//uplo == Lower

  }else{//side == Right
    int n1, n2;

    if(REG_SIZE(n))
      n1 = n2 = n/2;
    else{
      n1 = CLOSEST_REG_SIZE(n);
      n2 = n-n1;
    }

    if(uplo == HIPBLAS_FILL_MODE_UPPER){
      //Right / Upper / NoTrans
      if(trans == HIPBLAS_OP_N){
        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m, n2,
                                alpha, Ac+n1+n1*incA, incA,
                                       Bc+n1*incB, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = cublasXgemm(handle,
                                 HIPBLAS_OP_N, trans,
                                 m, n2, n1,
                                 alpha, Bc, incB,
                                        Ac+n1*incA, incA,
                                 &one,  Bc+n1*incB, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m, n1,
                                alpha, Ac, incA,
                                       Bc, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
      //Right / Upper / [Conj]Trans
      else{
        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m, n1,
                                alpha, Ac, incA,
                                       Bc, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = cublasXgemm(handle,
                                 HIPBLAS_OP_N, trans,
                                 m, n1, n2,
                                 alpha, Bc+n1*incB, incB,
                                        Ac+n1*incA, incA,
                                 &one,  Bc, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m, n2,
                                alpha, Ac+n1+n1*incA, incA,
                                       Bc+n1*incB, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
    }else{
      //Right / Lower / NoTrans
      if(trans == HIPBLAS_OP_N){
        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m, n1,
                                alpha, Ac, incA,
                                       Bc, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = cublasXgemm(handle,
                                 HIPBLAS_OP_N, trans,
                                 m, n1, n2,
                                 alpha, Bc+n1*incB, incB,
                                        Ac+n1, incA,
                                 &one,  Bc, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m, n2,
                                alpha, Ac+n1+n1*incA, incA,
                                       Bc+n1*incB, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
      //Right / Lower / [Conj]Trans
      else{
        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m, n2,
                                alpha, Ac+n1+n1*incA, incA,
                                       Bc+n1*incB, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = cublasXgemm(handle,
                                 HIPBLAS_OP_N, trans,
                                 m, n2, n1,
                                 alpha, Bc, incB,
                                        Ac+n1, incA,
                                 &one,  Bc+n1*incB, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrmm(handle,
                                side, uplo, trans, diag,
                                m, n1,
                                alpha, Ac, incA,
                                       Bc, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
    }

  }//side == Right

  check_error( hipStreamDestroy( inStream ), HIPBLAS_STATUS_INTERNAL_ERROR );
  check_error( hipEventDestroy( eDataIn ), HIPBLAS_STATUS_INTERNAL_ERROR);
  check_error( hipEventDestroy( eComp ), HIPBLAS_STATUS_INTERNAL_ERROR);
  return HIPBLAS_STATUS_SUCCESS;
}
*/
template<class T>
hipblasStatus_t kblasXtrmm_cpu(hipblasHandle_t handle, 
                              hipblasSideMode_t side, hipblasFillMode_t uplo,
                              hipblasOperation_t trans, hipblasDiagType_t diag,
                              int m, int n,
                              const T *alpha,
                              const T *h_A, int incA,
                                    T *h_B, int incB){
  //allocate memory on device
  T *d_A, *d_B;
  int Am, An, Bm, Bn;
  if ( side == HIPBLAS_SIDE_LEFT ) {
    Am = An = m;
  } else {
    Am = An = n;
  }
  Bm = m;
  Bn = n;

  /*check_error( hipHostRegister((void*)h_A, Am * An * sizeof(T), hipHostRegisterDefault), HIPBLAS_STATUS_INTERNAL_ERROR);
  check_error( hipHostRegister((void*)h_B, Bm * Bn * sizeof(T), hipHostRegisterDefault), HIPBLAS_STATUS_INTERNAL_ERROR);*/

  hipblasStatus_t status;
  int AsyncEngineCount, devID;
  check_error( hipGetDevice(&devID), HIPBLAS_STATUS_INTERNAL_ERROR);
  check_error( hipDeviceGetAttribute(&AsyncEngineCount, hipDeviceAttributeAsyncEngineCount, devID), HIPBLAS_STATUS_INTERNAL_ERROR);
  bool DO_INLINE_BOUT = AsyncEngineCount > 1;
  
  check_error( hipMalloc( (void**)&d_A, (Am*An)*sizeof(T) ), HIPBLAS_STATUS_ALLOC_FAILED);
  if(d_A == NULL) return HIPBLAS_STATUS_ALLOC_FAILED;
  check_error( hipMalloc( (void**)&d_B, (Bm*Bn)*sizeof(T) ), HIPBLAS_STATUS_ALLOC_FAILED);
  if(d_B == NULL) return HIPBLAS_STATUS_ALLOC_FAILED;
  
  //setup streams
  hipStream_t inStream, outStream;
  check_error( hipStreamCreateWithFlags( &inStream, hipStreamNonBlocking), HIPBLAS_STATUS_INTERNAL_ERROR );
  if(DO_INLINE_BOUT)
    check_error( hipStreamCreateWithFlags( &outStream, hipStreamNonBlocking), HIPBLAS_STATUS_INTERNAL_ERROR );
  
  //call cpu API trmm
  check_error( 
    (status = kblasXtrmm(handle, inStream, outStream,
                         side, uplo, trans,diag,
                         m, n,
                         alpha, h_A, incA, d_A,
                                h_B, incB, d_B,
                         false, DO_INLINE_BOUT, false)
    ), status);
  //sync streams
  if(DO_INLINE_BOUT){
    check_error( hipStreamSynchronize( outStream ), HIPBLAS_STATUS_INTERNAL_ERROR);
  }else{
    hipStream_t compStream;
    check_error( hipblasGetStream(handle, &compStream), HIPBLAS_STATUS_INTERNAL_ERROR);
    check_error( hipStreamSynchronize( compStream ), HIPBLAS_STATUS_INTERNAL_ERROR);
    check_error( status = hipblasGetMatrixAsync( m, n, sizeof(T), d_B, incB, h_B, incB, inStream), status);
  }
  //revoke streams
  check_error( hipStreamDestroy( inStream ), HIPBLAS_STATUS_INTERNAL_ERROR);
  if(DO_INLINE_BOUT)
    check_error( hipStreamDestroy( outStream ), HIPBLAS_STATUS_INTERNAL_ERROR);

  /*check_error( hipHostUnregister( (void*)h_A ), HIPBLAS_STATUS_INTERNAL_ERROR );
  check_error( hipHostUnregister( (void*)h_B ), HIPBLAS_STATUS_INTERNAL_ERROR );*/

  //free device memory
  //TODO should free aslo in case some other funtions above failed (don't just return)
  check_error( hipFree( d_A ), HIPBLAS_STATUS_INTERNAL_ERROR );
  check_error( hipFree( d_B ), HIPBLAS_STATUS_INTERNAL_ERROR );  
  check_error( hipGetLastError(), HIPBLAS_STATUS_EXECUTION_FAILED );
  return HIPBLAS_STATUS_SUCCESS;
}


//==============================================================================================

  /*
extern "C" {
int kblas_strmm_async(
  char side, char uplo, char trans, char diag,
  int m, int n,
  float alpha, const float *A, int incA,
                       float *B, int incB,
  hipStream_t    stream){

  return kblasXtrmm(
    side, uplo, trans, diag,
    m, n,
    alpha, A, incA,
           B, incB,
    stream);
}
int kblas_dtrmm_async(
  char side, char uplo, char trans, char diag,
  int m, int n,
  double alpha, const double *A, int incA,
                        double *B, int incB,
  hipStream_t    stream){

  return kblasXtrmm(
    side, uplo, trans, diag,
    m, n,
    alpha, A, incA,
           B, incB,
    stream);
}
int kblas_ctrmm_async(
  char side, char uplo, char trans, char diag,
  int m, int n,
  hipComplex alpha, const hipComplex *A, int incA,
                          hipComplex *B, int incB,
  hipStream_t    stream){

  return kblasXtrmm(
    side, uplo, trans, diag,
    m, n,
    alpha, A, incA,
           B, incB,
    stream);
}
int kblas_ztrmm_async(
  char side, char uplo, char trans, char diag,
  int m, int n,
  hipDoubleComplex alpha, const hipDoubleComplex *A, int incA,
                                hipDoubleComplex *B, int incB,
  hipStream_t    stream){

  return kblasXtrmm(
    side, uplo, trans, diag,
    m, n,
    alpha, A, incA,
           B, incB,
    stream);
}*/
  //==============================================================================================

#define kblasXtrmm_async_BODY {                                                                           \
  hipblasHandle_t cublas_handle;                                                                           \
  check_error( hipblasCreate(&cublas_handle), void() );                                                      \
  if( hipblasSetStream(cublas_handle, stream) != HIPBLAS_STATUS_SUCCESS ){                               \
    check_error( hipblasDestroy(cublas_handle), void());                                                  \
    return;                                                                                               \
  }                                                                                                       \
  hipblasSideMode_t  side_v2  = (side  == KBLAS_Left  ? HIPBLAS_SIDE_LEFT : HIPBLAS_SIDE_RIGHT);             \
  hipblasFillMode_t  uplo_v2  = (uplo  == KBLAS_Lower ? HIPBLAS_FILL_MODE_LOWER : HIPBLAS_FILL_MODE_UPPER);  \
  hipblasOperation_t trans_v2 = (trans == KBLAS_Trans ? HIPBLAS_OP_T : HIPBLAS_OP_N);                        \
  hipblasDiagType_t  diag_v2  = (diag  == KBLAS_Unit  ? HIPBLAS_DIAG_UNIT : HIPBLAS_DIAG_NON_UNIT);          \
                                                                                                          \
  check_error( kblasXtrmm(cublas_handle,                                                                  \
                          side_v2, uplo_v2, trans_v2, diag_v2,                                            \
                          m, n,                                                                           \
                          &alpha, A, lda,                                                                 \
                                  B, ldb), void());                                                         \
                                                                                                          \
  check_error( hipblasDestroy(cublas_handle), void());                                                    \
}

extern "C"{
void kblasStrmm_async(char side, char uplo, char trans, char diag,
                      int m, int n,
                      float alpha, const float *A, int lda,
                                         float *B, int ldb,
                      hipStream_t stream){
  kblasXtrmm_async_BODY
}

void kblasDtrmm_async(char side, char uplo, char trans, char diag,
                      int m, int n,
                      double alpha, const double *A, int lda,
                                          double *B, int ldb,
                      hipStream_t stream){
  kblasXtrmm_async_BODY
}
void kblasCtrmm_async(char side, char uplo, char trans, char diag,
                      int m, int n,
                      hipComplex alpha, const hipComplex *A, int lda,
                                             hipComplex *B, int ldb,
                      hipStream_t stream){
  kblasXtrmm_async_BODY
}
void kblasZtrmm_async(char side, char uplo, char trans, char diag,
                      int m, int n,
                      hipDoubleComplex alpha, const hipDoubleComplex *A, int lda,
                                                   hipDoubleComplex *B, int ldb,
                      hipStream_t stream){
  kblasXtrmm_async_BODY
}
}
//==============================================================================================

void kblasStrmm(char side, char uplo, char trans, char diag,
                int m, int n,
                float alpha, const float *A, int lda,
                                   float *B, int ldb){

  kblasStrmm_async(side, uplo, trans, diag,
                   m, n,
                   alpha, A, lda,
                          B, ldb,
                   0);
}

void kblasDtrmm(char side, char uplo, char trans, char diag,
                int m, int n,
                double alpha, const double *A, int lda,
                                    double *B, int ldb){

  kblasDtrmm_async(side, uplo, trans, diag,
                   m, n,
                   alpha, A, lda,
                          B, ldb,
                   0);
}
void kblasCtrmm(char side, char uplo, char trans, char diag,
                int m, int n,
                hipComplex alpha, const hipComplex *A, int lda,
                                       hipComplex *B, int ldb){

  kblasCtrmm_async(side, uplo, trans, diag,
                   m, n,
                   alpha, A, lda,
                          B, ldb,
                   0);

}
void kblasZtrmm(char side, char uplo, char trans, char diag,
                int m, int n,
                hipDoubleComplex alpha, const hipDoubleComplex *A, int lda,
                                             hipDoubleComplex *B, int ldb){

  kblasZtrmm_async(side, uplo, trans, diag,
                   m, n,
                   alpha, A, lda,
                          B, ldb,
                   0);
}
//==============================================================================================

hipblasStatus_t kblasStrmm(hipblasHandle_t handle,
                          hipblasSideMode_t side, hipblasFillMode_t uplo,
                          hipblasOperation_t trans, hipblasDiagType_t diag,
                          int m, int n,
                          const float *alpha,
                          const float *A, int lda,
                                float *B, int ldb){
  return kblasXtrmm(handle,
                    side, uplo, trans, diag,
                    m, n,
                    alpha, A, lda,
                           B, ldb);
}
hipblasStatus_t kblasDtrmm(hipblasHandle_t handle,
                          hipblasSideMode_t side, hipblasFillMode_t uplo,
                          hipblasOperation_t trans, hipblasDiagType_t diag,
                          int m, int n,
                          const double *alpha,
                          const double *A, int lda,
                                double *B, int ldb){
  return kblasXtrmm(handle,
                    side, uplo, trans, diag,
                    m, n,
                    alpha, A, lda,
                           B, ldb);
}
hipblasStatus_t kblasCtrmm(hipblasHandle_t handle,
                          hipblasSideMode_t side, hipblasFillMode_t uplo,
                          hipblasOperation_t trans, hipblasDiagType_t diag,
                          int m, int n,
                          const hipComplex *alpha,
                          const hipComplex *A, int lda,
                                hipComplex *B, int ldb){
  return kblasXtrmm(handle,
                    side, uplo, trans, diag,
                    m, n,
                    alpha, A, lda,
                           B, ldb);
}
hipblasStatus_t kblasZtrmm(hipblasHandle_t handle,
                          hipblasSideMode_t side, hipblasFillMode_t uplo,
                          hipblasOperation_t trans, hipblasDiagType_t diag,
                          int m, int n,
                          const hipDoubleComplex *alpha,
                          const hipDoubleComplex *A, int lda,
                                hipDoubleComplex *B, int ldb){
  return kblasXtrmm(handle,
                    side, uplo, trans, diag,
                    m, n,
                    alpha, A, lda,
                           B, ldb);
}


//==============================================================================================
hipblasStatus_t kblas_strmm(hipblasHandle_t handle,
                           hipblasSideMode_t side, hipblasFillMode_t uplo,
                           hipblasOperation_t trans, hipblasDiagType_t diag,
                           int m, int n,
                           const float *alpha,
                           const float *A, int lda,
                                 float *B, int ldb){
  return kblasXtrmm_cpu(handle,
                        side, uplo, trans, diag,
                        m, n,
                        alpha, A, lda,
                               B, ldb);
}

hipblasStatus_t kblas_dtrmm(hipblasHandle_t handle,
                           hipblasSideMode_t side, hipblasFillMode_t uplo,
                           hipblasOperation_t trans, hipblasDiagType_t diag,
                           int m, int n,
                           const double *alpha,
                           const double *A, int lda,
                                 double *B, int ldb){
  return kblasXtrmm_cpu(handle,
                        side, uplo, trans, diag,
                        m, n,
                        alpha, A, lda,
                               B, ldb);
}
hipblasStatus_t kblas_ctrmm(hipblasHandle_t handle,
                           hipblasSideMode_t side, hipblasFillMode_t uplo,
                           hipblasOperation_t trans, hipblasDiagType_t diag,
                           int m, int n,
                           const hipComplex *alpha,
                           const hipComplex *A, int lda,
                                 hipComplex *B, int ldb){
  return kblasXtrmm_cpu(handle,
                        side, uplo, trans, diag,
                        m, n,
                        alpha, A, lda,
                               B, ldb);
}
hipblasStatus_t kblas_ztrmm(hipblasHandle_t handle,
                           hipblasSideMode_t side, hipblasFillMode_t uplo,
                           hipblasOperation_t trans, hipblasDiagType_t diag,
                           int m, int n,
                           const hipDoubleComplex *alpha,
                           const hipDoubleComplex *A, int lda,
                                 hipDoubleComplex *B, int ldb){
  return kblasXtrmm_cpu(handle,
                        side, uplo, trans, diag,
                        m, n,
                        alpha, A, lda,
                               B, ldb);
}

//}


