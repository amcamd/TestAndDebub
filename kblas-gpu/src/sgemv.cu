/*
    -- KBLAS (version 1.0) --
       Ahmad Abdelfattah, Center of Extreme Computing
	   Hatem Ltaief, Supercomputing Laboratory
	   David Keyes, Center of Extreme Computing
	   King Abdullah University of Science and Technology (KAUST)
       June 2013
	   KBLAS is a subset of BLAS routines highly optimized for NVIDIA GPUs 
*/
/**
	-- Center of Extreme Computing and Supercomputing Laboratory
	-- Division of Applied Mathematics and Computational Science
	-- King Abdullah University of Science and Technology
	-- (C) Copyright 2013

	Redistribution  and  use  in  source and binary forms, with or without
	modification,  are  permitted  provided  that the following conditions
	are met:

	*	Redistributions  of  source  code  must  retain  the above copyright
		notice,  this  list  of  conditions  and  the  following  disclaimer.
	* 	Redistributions  in  binary  form must reproduce the above copyright
		notice,  this list of conditions and the following disclaimer in the
		documentation  and/or other materials provided with the distribution.
	* 	Neither  the  name of the University of Tennessee, Knoxville nor the
		names of its contributors may be used to endorse or promote products
		derived from this software without specific prior written permission.

	THIS  SOFTWARE  IS  PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
	''AS IS''  AND  ANY  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
	LIMITED  TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
	A  PARTICULAR  PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
	HOLDERS OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
	SPECIAL,  EXEMPLARY,  OR  CONSEQUENTIAL  DAMAGES  (INCLUDING,  BUT NOT
	LIMITED  TO,  PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
	DATA,  OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
	THEORY  OF  LIABILITY,  WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
	(INCLUDING  NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
	OF  THIS  SOFTWARE,  EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
**/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include "gemv_core.cuh"

#if(SM >= 30)

#define sgemvn_bs		(64)
#define sgemvn_ty		(8)
#define sgemvn_by		(8)

#define sgemvt_bs		(64)
#define sgemvt_ty		(4)
#define sgemvt_by		(4)

#else

#define sgemvn_bs		(32)
#define sgemvn_ty		(8)
#define sgemvn_by		(1)

#define sgemvt_bs		(32)
#define sgemvt_ty		(8)
#define sgemvt_by		(1)

#endif

extern "C"
int kblas_sscal_async(int n, float alpha, float *x, int incx, hipStream_t stream);

int kblas_sgemv_driver( char trans, int rows, int cols,
						float alpha, float *dA, int lda, 
						float *dX, int incx, 
						float  beta, float *dY, int incy, hipStream_t stream)
{
	if(trans == 'n' || trans == 'N')
	{
		// scaling with beta
		kblas_sscal_async(rows, beta, dY, incy, stream);

		int mod_r = rows % sgemvn_bs;
		int mod_c = cols % sgemvn_bs;	
		
		if(mod_r == 0)
		{
			if(mod_c == 0)
			{
				// special case
				int blocks = rows/sgemvn_bs;
				const int thread_x = sgemvn_bs;
				const int thread_y = sgemvn_ty; 
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, sgemvn_by);
				const int elements_per_thread = thread_x/(2*thread_y);
				gemvn_special<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy);
				//gemvn_special_<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy);
			}
			else
			{
				// generic case for columns only
				int blocks = rows/sgemvn_bs; 
				blocks += 1;	// dummy thread block
				const int thread_x = sgemvn_bs;
				const int thread_y = sgemvn_ty; 
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, sgemvn_by);
				const int elements_per_thread = thread_x/(2*thread_y);
				const int irregular_cols = mod_c % elements_per_thread;
				switch(irregular_cols)
				{
					/**
					 * The kernel for irregular dimensions has an extra template parameter.
				 	 * This parameter must be among the values listed in the switch-case statement below.
				 	 * The possible values are in the range 0 - (elements_per_thread-1)
				 	 * Make sure these values are updated whenever you change the configuration parameters.  
					**/	
					case  0: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  0><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  1: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  1><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  2: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  2><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  3: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  3><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  4: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  4><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  5: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  5><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  6: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  6><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  7: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  7><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  8: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  8><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					default: printf("SGEMV-N error: improper template parameter. Please read the inline documentation for this function. \n"); return -1;
					
				}
			}
		}
		else	// mod_r != 0
		{
			if(mod_c == 0)
			{
				// generic case for columns only
				int blocks = (rows/sgemvn_bs) + (mod_r != 0);
				const int thread_x = sgemvn_bs;
				const int thread_y = sgemvn_ty;
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, sgemvn_by);
				const int elements_per_thread = thread_x/(2*thread_y);
				gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread, 0><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c);
			}
			else
			{
				// generic case for rows and cols
				int blocks = (rows/sgemvn_bs) + (mod_r != 0);
				const int thread_x = sgemvn_bs;
				const int thread_y = sgemvn_ty;
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, sgemvn_by);
				const int elements_per_thread = thread_x/(2*thread_y);
				const int irregular_cols = mod_c % elements_per_thread;
				switch(irregular_cols)
				{
					/**
					 * The kernel for irregular dimensions has an extra template parameter.
				 	 * This parameter must be among the values listed in the switch-case statement below.
				 	 * The possible values are in the range 0 - (elements_per_thread-1)
				 	 * Make sure these values are updated whenever you change the configuration parameters.  
					**/	
					case  0: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  0><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  1: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  1><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  2: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  2><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  3: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  3><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  4: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  4><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  5: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  5><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  6: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  6><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  7: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  7><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  8: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  8><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					default: printf("SGEMV-N error: improper template parameter. Please read the inline documentation for this function. \n"); return -1;
					
				}
			}
		}
	}	// end of non-transpose case
	else if (trans == 't' || trans == 'T' || trans == 'c' || trans == 'C')
	{
		int conj;
		if(trans == 'c' || trans == 'C') conj = 1;
		else conj = 0;
		// scaling with beta
		kblas_sscal_async(cols, beta, dY, incy, stream);

		int mod_r = rows % sgemvt_bs;
		int mod_c = cols % sgemvt_bs;
		
		if(mod_c == 0)
		{
			if(mod_r == 0)
			{
				// special case
				int blocks = cols/sgemvt_bs;
				const int thread_x = sgemvt_bs;
				const int thread_y = sgemvt_ty;
				const int elements_per_thread = thread_x/(2*thread_y);
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, sgemvt_by);
				gemvt_special<float, sgemvt_bs, thread_x, thread_y, elements_per_thread><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, conj); 
			}
			else
			{
				// mod_r != 0
				int blocks = cols/sgemvt_bs;
				blocks += 1;	// dummy thread block
				const int thread_x = sgemvt_bs;
				const int thread_y = sgemvt_ty;
				const int elements_per_thread = thread_x/(2*thread_y);
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, sgemvt_by);
				gemvt_generic<float, sgemvt_bs, thread_x, thread_y, elements_per_thread, 0><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj);
			}
		}
		else	// mod_c != 0
		{
			int blocks = cols/sgemvt_bs + (mod_c != 0);
			const int thread_x = sgemvt_bs;
			const int thread_y = sgemvt_ty;
			const int elements_per_thread = thread_x/(2*thread_y);
			const int irregular_cols = mod_c % elements_per_thread;
			dim3 dimBlock(thread_x, thread_y);
			dim3 dimGrid(blocks, sgemvt_by);
			
			switch(irregular_cols)
			{
				/**
				  * The kernel for irregular dimensions has an extra template parameter.
				  * This parameter must be among the values listed in the switch-case statement below.
				  * The possible values are in the range 0 - (elements_per_thread-1)
				  * Make sure these values are updated whenever you change the configuration parameters.  
				**/	
				case  0: gemvt_generic<float, sgemvt_bs, thread_x, thread_y, elements_per_thread,  0><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				case  1: gemvt_generic<float, sgemvt_bs, thread_x, thread_y, elements_per_thread,  1><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				case  2: gemvt_generic<float, sgemvt_bs, thread_x, thread_y, elements_per_thread,  2><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				case  3: gemvt_generic<float, sgemvt_bs, thread_x, thread_y, elements_per_thread,  3><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				case  4: gemvt_generic<float, sgemvt_bs, thread_x, thread_y, elements_per_thread,  4><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				case  5: gemvt_generic<float, sgemvt_bs, thread_x, thread_y, elements_per_thread,  5><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				case  6: gemvt_generic<float, sgemvt_bs, thread_x, thread_y, elements_per_thread,  6><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				case  7: gemvt_generic<float, sgemvt_bs, thread_x, thread_y, elements_per_thread,  7><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				case  8: gemvt_generic<float, sgemvt_bs, thread_x, thread_y, elements_per_thread,  8><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				default: printf("SGEMV-T error: improper template parameter. Please read the inline documentation for this function. \n"); return -1;
			}
		}
	}
	else 
	{
		printf("SGEMV error: Unrecognized transpose mode %c \n", trans);
		return -1;
	}
	
	return 0;
}

extern "C"
int kblas_sgemv(char trans, int rows, int cols,
				float alpha, float *dA, int lda, 
				float *dX, int incx, 
				float  beta, float *dY, int incy)
{
	return kblas_sgemv_driver( trans, rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, 0);
}

extern "C"
int kblas_sgemv_async( 	char trans, int rows, int cols,
						float alpha, float *dA, int lda, 
						float *dX, int incx, 
						float  beta, float *dY, int incy,
						hipStream_t stream)
{
	return kblas_sgemv_driver( trans, rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, stream);
}
