#include "hip/hip_runtime.h"
/**
  - -* (C) Copyright 2013 King Abdullah University of Science and Technology
  Authors:
  Ali Charara (ali.charara@kaust.edu.sa)
  David Keyes (david.keyes@kaust.edu.sa)
  Hatem Ltaief (hatem.ltaief@kaust.edu.sa)

  Redistribution  and  use  in  source and binary forms, with or without
  modification,  are  permitted  provided  that the following conditions
  are met:

  * Redistributions  of  source  code  must  retain  the above copyright
  * notice,  this  list  of  conditions  and  the  following  disclaimer.
  * Redistributions  in  binary  form must reproduce the above copyright
  * notice,  this list of conditions and the following disclaimer in the
  * documentation  and/or other materials provided with the distribution.
  * Neither  the  name of the King Abdullah University of Science and
  * Technology nor the names of its contributors may be used to endorse
  * or promote products derived from this software without specific prior
  * written permission.
  *
  THIS  SOFTWARE  IS  PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
  ``AS IS''  AND  ANY  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
  LIMITED  TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
  A  PARTICULAR  PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
  HOLDERS OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
  SPECIAL,  EXEMPLARY,  OR  CONSEQUENTIAL  DAMAGES  (INCLUDING,  BUT NOT
  LIMITED  TO,  PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
  DATA,  OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
  THEORY  OF  LIABILITY,  WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
  (INCLUDING  NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
  OF  THIS  SOFTWARE,  EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
**/
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "hipblas.h"
#include "kblas.h"
#include "Xtr_common.ch"
#include "operators.h"

//==============================================================================================

hipblasStatus_t cublasXtrsm(hipblasHandle_t handle,
                           hipblasSideMode_t side, hipblasFillMode_t uplo,
                           hipblasOperation_t trans, hipblasDiagType_t diag,
                           int m, int n,
                           const float *alpha,
                           const float *A, int lda,
                           float *B, int ldb){
  hipblasStatus_t status;
  check_error( status = hipblasStrsm(handle,
                                    side, uplo, trans, diag,
                                    m, n,
                                    alpha, A, lda,
                                           B, ldb ), status);
  check_error( hipGetLastError(), HIPBLAS_STATUS_EXECUTION_FAILED );
  return HIPBLAS_STATUS_SUCCESS;
}
hipblasStatus_t cublasXtrsm(hipblasHandle_t handle,
                           hipblasSideMode_t side, hipblasFillMode_t uplo,
                           hipblasOperation_t trans, hipblasDiagType_t      diag,
                           int m, int n,
                           const double *alpha,
                           const double *A, int lda,
                                 double *B, int ldb){
  hipblasStatus_t status;
  check_error( status = hipblasDtrsm(handle,
                                    side, uplo, trans, diag,
                                    m, n,
                                    alpha, A, lda,
                                           B, ldb ), status);
  check_error( hipGetLastError(), HIPBLAS_STATUS_EXECUTION_FAILED );
  return HIPBLAS_STATUS_SUCCESS;
}
hipblasStatus_t cublasXtrsm (hipblasHandle_t handle,
                            hipblasSideMode_t side, hipblasFillMode_t uplo,
                            hipblasOperation_t trans, hipblasDiagType_t diag,
                            int m, int n,
                            const hipComplex *alpha,
                            const hipComplex *A, int lda,
                                  hipComplex *B, int ldb){
  hipblasStatus_t status;
  check_error( status = hipblasCtrsm(handle,
                                    side, uplo, trans, diag,
                                    m, n,
                                    alpha, A, lda,
                                           B, ldb ), status);
  check_error( hipGetLastError(), HIPBLAS_STATUS_EXECUTION_FAILED );
  return HIPBLAS_STATUS_SUCCESS;
}
hipblasStatus_t cublasXtrsm (hipblasHandle_t handle,
                            hipblasSideMode_t side, hipblasFillMode_t uplo,
                            hipblasOperation_t trans, hipblasDiagType_t diag,
                            int m, int n,
                            const hipDoubleComplex *alpha,
                            const hipDoubleComplex *A, int lda,
                                  hipDoubleComplex *B, int ldb){
  hipblasStatus_t status;
  check_error( status = hipblasZtrsm(handle,
                                    side, uplo, trans, diag,
                                    m, n,
                                    alpha, A, lda,
                                           B, ldb ), status);
  check_error( hipGetLastError(), HIPBLAS_STATUS_EXECUTION_FAILED );
  return HIPBLAS_STATUS_SUCCESS;
}

//==============================================================================================
#define WARP 32
#define WARP1 33
#define WARP2 34
#define tx threadIdx.x
#define ty threadIdx.y
//==============================================================================================
int kblas_trsm_ib_cublas = 128;
bool kblas_trsm_use_custom = 0;
int kblas_trsm_ib_data = 512;
#define SIMPLE_SIZE(n) ( ((n) < WARP) || ( ((n) % WARP == 0) && ( (n) <= kblas_trsm_ib_cublas ) ) )
#define SIMPLE_SIZE_DATA(n) ( (n) <= kblas_trsm_ib_data )
//==============================================================================================

//shuffle intrinsic is not supported before KEPLER
#if (SM >= 30)
template<typename T, int WARPS_PER_BLOCK, bool LOWER, bool TRANS, bool CONJG, bool UNIT>
__global__ void //__launch_bounds__(WARP * WARPS_PER_BLOCK)
trsm_mul32_L(int M, int N, T alpha, const T* __restrict__ A, int incA, T* B, int incB, int mb)
{
  const int A_COLS_PER_WARP = WARP / WARPS_PER_BLOCK;
  const bool forward = (LOWER != TRANS);

  //setup shared memory
  __shared__ T sA[WARP * WARP1];//strided to avoid bank conflicts

  int txyw = tx + ty * WARP1, txyiA = tx + ty * incA, txyiB = tx + ty * incB, jtxw;
  int l, c, r, startB = 0, i;
  T rB, s, rBj, a[4], b[4], *sAA, *BB;

  for(startB = 0; startB < N; startB += gridDim.x * WARPS_PER_BLOCK)
  {

    if( (blockIdx.x * WARPS_PER_BLOCK + startB) >= N)
      return;

    BB = B + (blockIdx.x * WARPS_PER_BLOCK + startB) * incB;

    //checking boundary case, the column indices of B this warp is computing
    //if not active, this warp will only participate in fetching A sub-matrices, will not compute
    bool active = ( (blockIdx.x * WARPS_PER_BLOCK + startB + ty) < N );

    for(c = (forward ? 0 : mb-1); (forward && c < mb) || (!forward && c >= 0); c += (forward ? 1 : -1))
    {
      s = make_zero<T>();

      for(r = (forward ? 0 : mb-1); (forward && r < c) || (!forward && r > c); r += (forward ? 1 : -1))
      {
        #pragma unroll
        for(l = 0; l < A_COLS_PER_WARP; l++){
          if(TRANS)
            //load A(r,c)
            sA[txyw + l * WARPS_PER_BLOCK * WARP1] = A[txyiA + WARP * (r + c * incA) + l * WARPS_PER_BLOCK * incA];
          else
            //load A(c,r)
            sA[txyw + l * WARPS_PER_BLOCK * WARP1] = A[txyiA + WARP * (c + r * incA) + l * WARPS_PER_BLOCK * incA];
        }
        //load B(r)
        if(active)
          rB = BB[txyiB + WARP * r];

        __syncthreads();
        if(active){
          //gemm A(r,c)/A(c,r) & B(r) onto B(c) held at s
          if(TRANS)
            sAA = sA + tx*WARP1;
          else
            sAA = sA + tx;
            #pragma unroll
            for(int j = 0; j < WARP; j+=4){
              if(TRANS){
                #pragma unroll
                for(i = 0; i < 4; i++)
                  a[i] = CONJG ? conjugate(sAA[j + i]) : sAA[j + i];
              }else{
                #pragma unroll
                for(i = 0; i < 4; i++)
                  a[i] = sAA[(j + i)*WARP1];
              }
              #pragma unroll
              for(i = 0; i < 4; i++)
                b[i] = shfl(rB, j + i);
              #pragma unroll
              for(i = 0; i < 4; i++)
                s = FMA( a[i], b[i], s );
            }
        }
        __syncthreads();
      }

      //load A(c,c) from global to shared mem
      #pragma unroll
      for(l = 0; l < A_COLS_PER_WARP; l++){
        sA[txyw + l * WARPS_PER_BLOCK * WARP1] = A[txyiA + WARP * c * (incA + 1) + l * WARPS_PER_BLOCK * incA];
      }

      //load B(c) into registers
      if(active){
        rB = BB[txyiB + WARP * c];
      }
      __syncthreads();
      if(active)
      {
        //perform trsm on shared mem
        if(!LOWER && TRANS)
          jtxw = tx * WARP1;
        else
        if(!LOWER && !TRANS)
          jtxw = tx         + (WARP - 1) * WARP1;
        else
        if(LOWER && TRANS)
          jtxw = tx * WARP1 + (WARP - 1);
        else
        if(LOWER && !TRANS)
          jtxw = tx;

        #pragma unroll
        for(int j = (forward ? 0 : WARP-1); (forward && (j < WARP)) || (!forward && (j >= 0)); j += (forward ? 1 : -1)){
          if(j == tx){
            rB = FMA(alpha, rB, -s);//TODO
            if(!UNIT){
              a[0] = (TRANS && CONJG) ? conjugate(sA[tx * WARP2]) : sA[tx * WARP2];//diagonal element
              rB = rB / a[0];//TODO
            }
          }
          rBj = shfl(rB, j);

          if( (forward && (j < tx)) || (!forward && (j > tx)) ){
            a[0] = (TRANS && CONJG) ? conjugate(sA[jtxw]) : sA[jtxw];
            s = FMA(a[0], rBj, s);
          }
          jtxw += (TRANS ? 1 : WARP1) * (forward ? 1 : -1);
        }

        //store back B(c) to global mem
        BB[txyiB + WARP * c] = rB;
      }
      __syncthreads();
    }
  }
}


//==============================================================================================
#define TRSM_NUM_VARIANTS 4
#define TRSM_kernel_variants(__WPB)                  \
        trsm_mul32_L<T, __WPB,  true, false, false, false>, \
        trsm_mul32_L<T, __WPB,  true,  true, false, false>, \
        trsm_mul32_L<T, __WPB, false, false, false, false>, \
        trsm_mul32_L<T, __WPB, false,  true, false, false>
        /*,TODO
        trsm_mul32_R<T, WARPS_PER_BLOCK, B_COLS_PER_WARP,  true, false, false>,
        trsm_mul32_R<T, WARPS_PER_BLOCK, B_COLS_PER_WARP,  true,  true, false>,
        trsm_mul32_R<T, WARPS_PER_BLOCK, B_COLS_PER_WARP, false, false, false>,
        trsm_mul32_R<T, WARPS_PER_BLOCK, B_COLS_PER_WARP, false,  true, false>*/
template<class T>
hipblasStatus_t Xtrsm(hipblasHandle_t handle,
                     hipblasSideMode_t side, hipblasFillMode_t uplo,
                     hipblasOperation_t trans, hipblasDiagType_t diag,
                     int m, int n,
                     const T *alpha,
                     const T *A, int incA,
                           T *B, int incB){
  
  //handle odd cases with cublas
  if(  (*alpha == make_zero<T>())
    || (!kblas_trsm_use_custom)
    || (side == HIPBLAS_SIDE_LEFT && m < WARP)
    || (side == HIPBLAS_SIDE_RIGHT/* && n < WARP*/))//TODO
  {
    return cublasXtrsm(handle,
                       side, uplo, trans, diag,
                       m, n,
                       alpha, A, incA,
                              B, incB );
  }

  typedef void (*trsm_kernels_type)(int M, int N, T alpha, const T* A, int incA, T* B, int incB, int mb);

  #define WARPS_PER_BLOCK 8
  #define B_COLS_PER_WARP 1

  trsm_kernels_type trsm_kernels[TRSM_NUM_VARIANTS] = {// T, WARPS_PER_BLOCK, LOWER, TRANS, CONJG, UNIT
    TRSM_kernel_variants(WARPS_PER_BLOCK)
  };

  hipStream_t curStream;
  hipblasStatus_t status;
  
  check_error( status = hipblasGetStream( handle, &curStream ), status);

  if( ((side == HIPBLAS_SIDE_LEFT) && (m % WARP == 0)) /*|| ((side == HIPBLAS_SIDE_RIGHT) && (n % WARP == 0))*/ )//TODO
  {
    int func_idx = /*4*(side == HIPBLAS_SIDE_RIGHT) + */2*(uplo == HIPBLAS_FILL_MODE_UPPER) + (trans != HIPBLAS_OP_N);// + (diag == HIPBLAS_DIAG_UNIT);TODO
    dim3 blockDim( WARP, WARPS_PER_BLOCK );
    dim3 gridDim(
      (side == HIPBLAS_SIDE_LEFT) * (n / (WARPS_PER_BLOCK * B_COLS_PER_WARP) + (n % (WARPS_PER_BLOCK * B_COLS_PER_WARP) > 0))
      /*+TODO
      (side == HIPBLAS_SIDE_RIGHT) * (m / (WARPS_PER_BLOCK * B_COLS_PER_WARP) + (m % (WARPS_PER_BLOCK * B_COLS_PER_WARP) > 0))*/
      , 1);
    int mb = (side == HIPBLAS_SIDE_LEFT) * m / WARP /*+ (side == HIPBLAS_SIDE_RIGHT) * n / WARP*/;//TODO
    trsm_kernels[func_idx]<<< gridDim, blockDim, 0, curStream>>> (m, n, *alpha, A, incA, B, incB, mb);
    check_error( hipGetLastError(),  HIPBLAS_STATUS_EXECUTION_FAILED );
  }else{
    //error: we should not reach this case
    return HIPBLAS_STATUS_INTERNAL_ERROR;
  }
  return HIPBLAS_STATUS_SUCCESS;
}

#else

template<class T>
hipblasStatus_t Xtrsm(hipblasHandle_t handle,
                     hipblasSideMode_t side, hipblasFillMode_t uplo,
                     hipblasOperation_t trans, hipblasDiagType_t diag,
                     int m, int n,
                     const T *alpha,
                     const T *A, int incA,
                     T *B, int incB){
  return cublasXtrsm( handle,
                      side, uplo, trans, diag,
                      m, n,
                      alpha, A, incA,
                             B, incB );
}

#endif
//==============================================================================================
template<typename T>
hipblasStatus_t kblasXtrsm(hipblasHandle_t handle,
                          hipblasSideMode_t side, hipblasFillMode_t uplo,
                          hipblasOperation_t trans, hipblasDiagType_t diag,
                          int m, int n,
                          const T *alpha,
                          const T *A, int incA,
                                T *B, int incB)
{
  T one = make_one<T>();
  T mone = make_zero<T>() - one;
  T mInvAlpha = mone / *alpha;
  hipblasStatus_t status;

  if( (*alpha == make_zero<T>())//TODO
   || ( (side == HIPBLAS_SIDE_LEFT) && (SIMPLE_SIZE(m)) )
   || ( (side == HIPBLAS_SIDE_RIGHT) && (SIMPLE_SIZE(n)) ) ){
    return Xtrsm(handle,
                 side, uplo, trans, diag,
                 m, n,
                 alpha, A, incA,
                        B, incB );
  }
  else
  if(side == HIPBLAS_SIDE_LEFT){

    int m1, m2;
    if(REG_SIZE(m))
      m1 = m2 = m/2;
    else{
      m1 = CLOSEST_REG_SIZE(m);
      m2 = m-m1;
    }

    if(uplo == HIPBLAS_FILL_MODE_UPPER){

      //Left / Upper / NoTrans
      if(trans == HIPBLAS_OP_N){
        if((status = kblasXtrsm(handle,
                                side, uplo, trans, diag,
                                m2, n,
                                alpha, A+m1+m1*incA, incA,
                                       B+m1, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = cublasXgemm(handle,
                                 trans, HIPBLAS_OP_N,
                                 m1, n, m2,
                                 &mone, A+m1*incA, incA,
                                        B+m1, incB,
                                 alpha, B, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrsm(handle,
                                side, uplo, trans, diag,
                                m1, n,
                                &one, A, incA,
                                      B, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
      //Left / Upper / [Conj]Trans
      else{
        if((status = kblasXtrsm(handle,
                                side, uplo, trans, diag,
                                m1, n,
                                alpha, A, incA,
                                       B, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = cublasXgemm(handle,
                                 trans, HIPBLAS_OP_N,
                                 m2, n, m1,
                                 &mone, A+m1*incA, incA,
                                        B, incB,
                                 alpha, B+m1, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrsm(handle,
                                side, uplo, trans, diag,
                                m2, n,
                                &one, A+m1+m1*incA, incA,
                                      B+m1, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
    }else{//uplo == KBLAS_Lower

      //Left / Lower / NoTrans
      if(trans == HIPBLAS_OP_N){
        if((status = kblasXtrsm(handle,
                                side, uplo, trans, diag,
                                m1, n,
                                alpha, A, incA,
                                       B, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = cublasXgemm(handle,
                                 trans, HIPBLAS_OP_N,
                                 m2, n, m1,
                                 &mone, A+m1, incA,
                                        B, incB,
                                 alpha, B+m1, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrsm(handle,
                                side, uplo, trans, diag,
                                m2, n,
                                &one, A+m1+m1*incA, incA,
                                      B+m1, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
      //Left / Lower / [Conj]Trans
      else{//transa == KBLAS_Trans

        if((status = kblasXtrsm(handle,
                                side, uplo, trans, diag,
                                m2, n,
                                alpha, A+m1+m1*incA, incA,
                                       B+m1, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = cublasXgemm(handle,
                                 trans, HIPBLAS_OP_N,
                                 m1, n, m2,
                                 &mone, A+m1, incA,
                                        B+m1, incB,
                                 alpha, B, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrsm(handle,
                                side, uplo, trans, diag,
                                m1, n,
                                &one, A, incA,
                                      B, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }//transa == KBLAS_Trans
    }
  }
  else{//side == KBLAS_Right
    int n1, n2;

    if(REG_SIZE(n))
      n1 = n2 = n/2;
    else{
      n1 = CLOSEST_REG_SIZE(n);
      n2 = n-n1;
    }

    if(uplo == HIPBLAS_FILL_MODE_UPPER){
      //Right / Upper / NoTrans
      if(trans == HIPBLAS_OP_N){
        if((status = kblasXtrsm(handle,
                                side, uplo, trans, diag,
                                m, n1,
                                alpha, A, incA,
                                       B, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = cublasXgemm(handle,
                                 HIPBLAS_OP_N, trans,
                                 m, n2, n1,
                                 &mone, B, incB,
                                        A+n1*incA, incA,
                                 alpha, B+n1*incB, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrsm(handle,
                                side, uplo, trans, diag,
                                m, n2,
                                &one, A+n1+n1*incA, incA,
                                      B+n1*incB, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
      //Right / Upper / [Conj]Trans
      else{
        if((status = kblasXtrsm(handle,
                                side, uplo, trans, diag,
                                m, n2,
                                alpha, A+n1+n1*incA, incA,
                                       B+n1*incB, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = cublasXgemm(handle,
                                 HIPBLAS_OP_N, trans,
                                 m, n1, n2,
                                 &mInvAlpha, B+n1*incB, incB,
                                             A+n1*incA, incA,
                                 &one,       B, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrsm(handle,
                                side, uplo, trans, diag,
                                m, n1,
                                alpha, A, incA,
                                       B, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
    }
    else{
      //Right / Lower / NoTrans
      if(trans == HIPBLAS_OP_N){
        if((status = kblasXtrsm(handle,
                                side, uplo, trans, diag,
                                m, n2,
                                alpha, A+n1+n1*incA, incA,
                                       B+n1*incB, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = cublasXgemm(handle,
                                 HIPBLAS_OP_N, trans,
                                 m, n1, n2,
                                 &mone, B+n1*incB, incB,
                                        A+n1, incA,
                                 alpha, B, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrsm(handle,
                                side, uplo, trans, diag,
                                m, n1,
                                &one, A, incA,
                                      B, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
      //Right / Lower / [Conj]Trans
      else{
        if((status = kblasXtrsm(handle,
                                side, uplo, trans, diag,
                                m, n1,
                                alpha, A, incA,
                                       B, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = cublasXgemm(handle,
                                 HIPBLAS_OP_N, trans,
                                 m, n2, n1,
                                 &mInvAlpha, B, incB,
                                             A+n1, incA,
                                 &one,       B+n1*incB, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrsm(handle,
                                side, uplo, trans, diag,
                                m, n2,
                                alpha, A+n1+n1*incA, incA,
                                       B+n1*incB, incB
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
    }

  }//side == Right

  return HIPBLAS_STATUS_SUCCESS;
}

//==============================================================================================
template<typename T>
hipblasStatus_t kblasXtrsm(hipblasHandle_t handle, hipStream_t &strIn, hipStream_t &strOut,
                          hipblasSideMode_t side, hipblasFillMode_t uplo,
                          hipblasOperation_t trans, hipblasDiagType_t diag,
                          int m, int n,
                          const T *alpha,
                          const T *h_A, int incA, T* d_A,
                                T *h_B, int incB, T* d_B,
                          bool BIsIn, bool getBOut, bool AIsIn)
{
  T one = make_one<T>();
  T mone = make_zero<T>() - one;
  T mInvAlpha = mone / *alpha;
  hipblasStatus_t status;
  hipblasOperation_t noTrans = HIPBLAS_OP_N;//Trans = HIPBLAS_OP_T,
  
  hipEvent_t eDataIn, eComp;
  check_error( hipEventCreateWithFlags(&eDataIn, hipEventDisableTiming), HIPBLAS_STATUS_EXECUTION_FAILED);
  check_error( hipEventCreateWithFlags(&eComp, hipEventDisableTiming), HIPBLAS_STATUS_EXECUTION_FAILED);
  hipStream_t strComp;
  check_error( hipblasGetStream(handle, &strComp), HIPBLAS_STATUS_INTERNAL_ERROR);

  if( (*alpha == make_zero<T>())//TODO
   || ( (side == HIPBLAS_SIDE_LEFT) && (SIMPLE_SIZE(m)) )
   || ( (side == HIPBLAS_SIDE_RIGHT) && (SIMPLE_SIZE(n)) ) ){

    int Am = (side == HIPBLAS_SIDE_LEFT) ? m : n;
    //if B is not already in, copy in B block
    if(!BIsIn)
      check_error( status = hipblasSetMatrixAsync( m, n, sizeof(T), h_B, incB, d_B, incB, strIn ), status);
    //copy in A block
    if(!AIsIn)
      check_error( status = hipblasSetMatrixAsync( Am, Am, sizeof(T), h_A, incA, d_A, incA, strIn ), status);
    //wait for data to arrive
    if(!AIsIn || !BIsIn){
      check_error( hipEventRecord(eDataIn, strIn), HIPBLAS_STATUS_INTERNAL_ERROR);
      check_error( hipStreamWaitEvent(strComp, eDataIn, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
    }
    if( (status = Xtrsm(handle,
                        side, uplo, trans, diag,
                        m, n,
                        alpha, d_A, incA,
                               d_B, incB ) ) != HIPBLAS_STATUS_SUCCESS ) return status;

    //if stream is done computing and getBOut, copy B back.
    if(getBOut){
      check_error( hipEventRecord(eComp, strComp), HIPBLAS_STATUS_INTERNAL_ERROR);
      check_error( hipStreamWaitEvent(strOut, eComp, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
      check_error( status = hipblasGetMatrixAsync( m, n, sizeof(T), d_B, incB, h_B, incB, strOut), status);
    }
  }
  else
  if(side == HIPBLAS_SIDE_LEFT){

    int m1, m2;
    if(REG_SIZE(m))
      m1 = m2 = m/2;
    else{
      m1 = CLOSEST_REG_SIZE(m);
      m2 = m-m1;
    }

    if( (!AIsIn && SIMPLE_SIZE_DATA(m)) || (!BIsIn && SIMPLE_SIZE_DATA(m)) ){
      if( (!AIsIn && SIMPLE_SIZE_DATA(m)) ){
        check_error( status = hipblasSetMatrixAsync( m, m, sizeof(T), h_A, incA, d_A, incA, strIn), status);
        AIsIn = true;
      }
      if( (!BIsIn && SIMPLE_SIZE_DATA(m)) ){
        check_error( status = hipblasSetMatrixAsync( m, n, sizeof(T), h_B, incB, d_B, incB, strIn), status);
        BIsIn = true;
      }
      //wait for data to arrive
      check_error( hipEventRecord(eDataIn, strIn), HIPBLAS_STATUS_INTERNAL_ERROR);
      check_error( hipStreamWaitEvent(strComp, eDataIn, 0), HIPBLAS_STATUS_INTERNAL_ERROR);      
    }

    if(uplo == HIPBLAS_FILL_MODE_UPPER){

      //Left / Upper / NoTrans
      if(trans == HIPBLAS_OP_N){
        if((status = kblasXtrsm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m2, n,
                                alpha, h_A+m1+m1*incA, incA, d_A+m1+m1*incA,
                                       h_B+m1, incB, d_B+m1,
                                BIsIn, false, AIsIn 
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
        //if stream is done computing and getBOut, copy B back.
        if(getBOut){
          check_error( hipEventRecord(eComp, strComp), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strOut, eComp, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( status = hipblasGetMatrixAsync( m2, n, sizeof(T), d_B+m1, incB, h_B+m1, incB, strOut), status);
        }

        //prepare needed data
        if(!AIsIn || !BIsIn){
          //if B is not already in, copy B block
          if(!BIsIn){
            check_error( status = hipblasSetMatrixAsync( m1, n, sizeof(T), h_B, incB, d_B, incB, strIn), status);
            BIsIn = true;
          }
          //copy in A block
          if(!AIsIn)
            check_error( status = hipblasSetMatrixAsync( m1, m2, sizeof(T), h_A+m1*incA, incA, d_A+m1*incA, incA, strIn), status);
          //wait for data to arrive
          check_error( hipEventRecord(eDataIn, strIn), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strComp, eDataIn, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
        }
        if((status = cublasXgemm(handle,
                                 trans, noTrans,
                                 m1, n, m2,
                                 &mone, d_A+m1*incA, incA,
                                        d_B+m1, incB,
                                 alpha, d_B, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrsm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m1, n,
                                &one, h_A, incA, d_A,
                                      h_B, incB, d_B,
                                BIsIn, getBOut, AIsIn
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
      //Left / Upper / [Conj]Trans
      else{
        if((status = kblasXtrsm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m1, n,
                                alpha, h_A, incA, d_A,
                                       h_B, incB, d_B,
                                BIsIn, false, AIsIn 
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
        //if stream is done computing and getBOut, copy B back.
        if(getBOut){
          check_error( hipEventRecord(eComp, strComp), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strOut, eComp, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( status = hipblasGetMatrixAsync( m1, n, sizeof(T), d_B, incB, h_B, incB, strOut), status);
        }

        //prepare needed data
        if(!AIsIn || !BIsIn){
          //if B is not already in, copy B block
          if(!BIsIn){
            check_error( status = hipblasSetMatrixAsync( m2, n, sizeof(T), h_B+m1, incB, d_B+m1, incB, strIn), status);
            BIsIn = true;
          }
          //copy in A block
          if(!AIsIn)
            check_error( status = hipblasSetMatrixAsync( m1, m2, sizeof(T), h_A+m1*incA, incA, d_A+m1*incA, incA, strIn), status);
          //wait for data to arrive
          check_error( hipEventRecord(eDataIn, strIn), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strComp, eDataIn, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
        }
        if((status = cublasXgemm(handle,
                                 trans, noTrans,
                                 m2, n, m1,
                                 &mone, d_A+m1*incA, incA,
                                        d_B, incB,
                                 alpha, d_B+m1, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrsm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m2, n,
                                &one, h_A+m1+m1*incA, incA, d_A+m1+m1*incA, 
                                      h_B+m1, incB, d_B+m1,  
                                BIsIn, getBOut, AIsIn
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
    }else{//uplo == KBLAS_Lower

      //Left / Lower / NoTrans
      if(trans == HIPBLAS_OP_N){
        if((status = kblasXtrsm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m1, n,
                                alpha, h_A, incA, d_A, 
                                       h_B, incB, d_B,
                                BIsIn, false, AIsIn 
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
        //if stream is done computing and getBOut, copy B back.
        if(getBOut){
          check_error( hipEventRecord(eComp, strComp), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strOut, eComp, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( status = hipblasGetMatrixAsync( m1, n, sizeof(T), d_B, incB, h_B, incB, strOut), status);
        }

        //prepare needed data
        if(!AIsIn || !BIsIn){
          //if B is not already in, copy B block
          if(!BIsIn){
            check_error( status = hipblasSetMatrixAsync( m2, n, sizeof(T), h_B+m1, incB, d_B+m1, incB, strIn), status);
            BIsIn = true;
          }
          //copy in A block
          if(!AIsIn)
            check_error( status = hipblasSetMatrixAsync( m2, m1, sizeof(T), h_A+m1, incA, d_A+m1, incA, strIn), status);
          //wait for data to arrive
          check_error( hipEventRecord(eDataIn, strIn), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strComp, eDataIn, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
        }

        if((status = cublasXgemm(handle,
                                 trans, noTrans,
                                 m2, n, m1,
                                 &mone, d_A+m1, incA,
                                        d_B, incB,
                                 alpha, d_B+m1, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrsm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m2, n,
                                &one, h_A+m1+m1*incA, incA, d_A+m1+m1*incA,
                                      h_B+m1, incB, d_B+m1, 
                                BIsIn, getBOut, AIsIn
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
      //Left / Lower / [Conj]Trans
      else{//transa == KBLAS_Trans

        if((status = kblasXtrsm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m2, n,
                                alpha, h_A+m1+m1*incA, incA, d_A+m1+m1*incA, 
                                       h_B+m1, incB, d_B+m1, 
                                BIsIn, false, AIsIn 
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
        //if stream is done computing and getBOut, copy B back.
        if(getBOut){
          check_error( hipEventRecord(eComp, strComp), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strOut, eComp, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( status = hipblasGetMatrixAsync( m2, n, sizeof(T), d_B+m1, incB, h_B+m1, incB, strOut), status);
        }

        //prepare needed data
        if(!AIsIn || !BIsIn){
          //if B is not already in, copy B block
          if(!BIsIn){
            check_error( status = hipblasSetMatrixAsync( m1, n, sizeof(T), h_B, incB, d_B, incB, strIn), status);
            BIsIn = true;
          }
          //copy in A block
          if(!AIsIn)
            check_error( status = hipblasSetMatrixAsync( m1, m2, sizeof(T), h_A+m1, incA, d_A+m1, incA, strIn), status);
          //wait for data to arrive
          check_error( hipEventRecord(eDataIn, strIn), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strComp, eDataIn, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
        }

        if((status = cublasXgemm(handle,
                                 trans, noTrans,
                                 m1, n, m2,
                                 &mone, d_A+m1, incA,
                                        d_B+m1, incB,
                                 alpha, d_B, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrsm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m1, n,
                                &one, h_A, incA, d_A,
                                      h_B, incB, d_B,
                                BIsIn, getBOut, AIsIn
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }//transa == KBLAS_Trans
    }
  }
  else{//side == KBLAS_Right
    int n1, n2;

    if(REG_SIZE(n))
      n1 = n2 = n/2;
    else{
      n1 = CLOSEST_REG_SIZE(n);
      n2 = n-n1;
    }

    if(uplo == HIPBLAS_FILL_MODE_UPPER){
      //Right / Upper / NoTrans
      if(trans == noTrans){
        if((status = kblasXtrsm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m, n1,
                                alpha, h_A, incA, d_A,
                                       h_B, incB, d_B,
                                BIsIn, false, AIsIn 
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
        //if stream is done computing and getBOut, copy B back.
        if(getBOut){
          check_error( hipEventRecord(eComp, strComp), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strOut, eComp, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( status = hipblasGetMatrixAsync( m, n1, sizeof(T), d_B, incB, h_B, incB, strOut), status);
        }

        //prepare needed data
        if(!AIsIn || !BIsIn){
          //if B is not already in, copy B block
          if(!BIsIn){
            check_error( status = hipblasSetMatrixAsync( m, n2, sizeof(T), h_B+n1*incB, incB, d_B+n1*incB, incB, strIn), status);
            BIsIn = true;
          }
          //copy in A block
          if(!AIsIn)
            check_error( status = hipblasSetMatrixAsync( n1, n2, sizeof(T), h_A+n1*incA, incA, d_A+n1*incA, incA, strIn), status);
          //wait for data to arrive
          check_error( hipEventRecord(eDataIn, strIn), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strComp, eDataIn, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
        }

        if((status = cublasXgemm(handle,
                                 noTrans, trans,
                                 m, n2, n1,
                                 &mone, d_B, incB,
                                        d_A+n1*incA, incA,
                                 alpha, d_B+n1*incB, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrsm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m, n2,
                                &one, h_A+n1+n1*incA, incA, d_A+n1+n1*incA,
                                      h_B+n1*incB, incB, d_B+n1*incB,
                                BIsIn, getBOut, AIsIn
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
      //Right / Upper / [Conj]Trans
      else{
        if((status = kblasXtrsm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m, n2,
                                alpha, h_A+n1+n1*incA, incA, d_A+n1+n1*incA,
                                       h_B+n1*incB, incB, d_B+n1*incB,
                                BIsIn, false, AIsIn 
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
        //if stream is done computing and getBOut, copy B back.
        if(getBOut){
          check_error( hipEventRecord(eComp, strComp), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strOut, eComp, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( status = hipblasGetMatrixAsync( m, n2, sizeof(T), d_B+n1*incB, incB, h_B+n1*incB, incB, strOut), status);
        }

        //prepare needed data
        if(!AIsIn || !BIsIn){
          //if B is not already in, copy B block
          if(!BIsIn){
            check_error( status = hipblasSetMatrixAsync( m, n1, sizeof(T), h_B, incB, d_B, incB, strIn), status);
            BIsIn = true;
          }
          //copy in A block
          if(!AIsIn)
            check_error( status = hipblasSetMatrixAsync( n1, n2, sizeof(T), h_A+n1*incA, incA, d_A+n1*incA, incA, strIn), status);
          //wait for data to arrive
          check_error( hipEventRecord(eDataIn, strIn), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strComp, eDataIn, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
        }

        if((status = cublasXgemm(handle,
                                 noTrans, trans,
                                 m, n1, n2,
                                 &mInvAlpha, d_B+n1*incB, incB,
                                             d_A+n1*incA, incA,
                                 &one,       d_B, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrsm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m, n1,
                                alpha, h_A, incA, d_A,
                                       h_B, incB, d_B,
                                BIsIn, getBOut, AIsIn
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
    }
    else{
      //Right / Lower / NoTrans
      if(trans == HIPBLAS_OP_N){
        if((status = kblasXtrsm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m, n2,
                                alpha, h_A+n1+n1*incA, incA, d_A+n1+n1*incA,
                                       h_B+n1*incB, incB, d_B+n1*incB,
                                BIsIn, false, AIsIn 
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
        //if stream is done computing and getBOut, copy B back.
        if(getBOut){
          check_error( hipEventRecord(eComp, strComp), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strOut, eComp, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( status = hipblasGetMatrixAsync( m, n2, sizeof(T), d_B+n1*incB, incB, h_B+n1*incB, incB, strOut), status);
        }

        //prepare needed data
        if(!AIsIn || !BIsIn){
          //if B is not already in, copy B block
          if(!BIsIn){
            check_error( status = hipblasSetMatrixAsync( m, n1, sizeof(T), h_B, incB, d_B, incB, strIn), status);
            BIsIn = true;
          }
          //copy in A block
          if(!AIsIn)
            check_error( status = hipblasSetMatrixAsync( n2, n1, sizeof(T), h_A+n1, incA, d_A+n1, incA, strIn), status);
          //wait for data to arrive
          check_error( hipEventRecord(eDataIn, strIn), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strComp, eDataIn, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
        }

        if((status = cublasXgemm(handle,
                                 noTrans, trans,
                                 m, n1, n2,
                                 &mone, d_B+n1*incB, incB,
                                        d_A+n1, incA,
                                 alpha, d_B, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrsm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m, n1,
                                &one, h_A, incA, d_A,
                                      h_B, incB, d_B,
                                BIsIn, getBOut, AIsIn
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
      //Right / Lower / [Conj]Trans
      else{
        if((status = kblasXtrsm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m, n1,
                                alpha, h_A, incA, d_A,
                                       h_B, incB, d_B,
                                BIsIn, false, AIsIn 
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
        //if stream is done computing and getBOut, copy B back.
        if(getBOut){
          check_error( hipEventRecord(eComp, strComp), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strOut, eComp, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( status = hipblasGetMatrixAsync( m, n1, sizeof(T), d_B, incB, h_B, incB, strOut), status);
        }

        //prepare needed data
        if(!AIsIn || !BIsIn){
          //if B is not already in, copy B block
          if(!BIsIn){
            check_error( status = hipblasSetMatrixAsync( m, n2, sizeof(T), h_B+n1*incB, incB, d_B+n1*incB, incB, strIn), status);
            BIsIn = true;
          }
          //copy in A block
          if(!AIsIn)
            check_error( status = hipblasSetMatrixAsync( n1, n2, sizeof(T), h_A+n1, incA, d_A+n1, incA, strIn), status);
          //wait for data to arrive
          check_error( hipEventRecord(eDataIn, strIn), HIPBLAS_STATUS_INTERNAL_ERROR);
          check_error( hipStreamWaitEvent(strComp, eDataIn, 0), HIPBLAS_STATUS_INTERNAL_ERROR);
        }

        if((status = cublasXgemm(handle,
                                 noTrans, trans,
                                 m, n2, n1,
                                 &mInvAlpha, d_B, incB,
                                             d_A+n1, incA,
                                 &one,       d_B+n1*incB, incB
                                 )) != HIPBLAS_STATUS_SUCCESS) return status;

        if((status = kblasXtrsm(handle, strIn, strOut,
                                side, uplo, trans, diag,
                                m, n2,
                                alpha, h_A+n1+n1*incA, incA, d_A+n1+n1*incA,
                                       h_B+n1*incB, incB, d_B+n1*incB,
                                BIsIn, getBOut, AIsIn
                                )) != HIPBLAS_STATUS_SUCCESS) return status;
      }
    }

  }//side == Right
  
  
  check_error( hipEventDestroy( eDataIn ), HIPBLAS_STATUS_INTERNAL_ERROR);
  check_error( hipEventDestroy( eComp ), HIPBLAS_STATUS_INTERNAL_ERROR);
  check_error( hipGetLastError(), HIPBLAS_STATUS_EXECUTION_FAILED );
  return HIPBLAS_STATUS_SUCCESS;
}

//==============================================================================================
//#define DO_INLINE_BOUT 0
template<class T>
hipblasStatus_t kblasXtrsm_cpu(hipblasHandle_t handle,
                              hipblasSideMode_t side, hipblasFillMode_t uplo,
                              hipblasOperation_t trans, hipblasDiagType_t diag,
                              int m, int n,
                              const T *alpha,
                              const T *h_A, int incA,
                                    T *h_B, int incB){
  //allocate memory on device
  T *d_A, *d_B;
  int Am, An, Bm, Bn;
  if ( side == HIPBLAS_SIDE_LEFT ) {
    Am = An = m;
  } else {
    Am = An = n;
  }
  Bm = m;
  Bn = n;

  /*check_error( hipHostRegister((void*)h_A, Am * An * sizeof(T), hipHostRegisterDefault), HIPBLAS_STATUS_INTERNAL_ERROR);
  check_error( hipHostRegister((void*)h_B, Bm * Bn * sizeof(T), hipHostRegisterDefault), HIPBLAS_STATUS_INTERNAL_ERROR);*/

  hipblasStatus_t status;
  //*
  int AsyncEngineCount, devID;
  check_error( hipGetDevice(&devID), HIPBLAS_STATUS_INTERNAL_ERROR);
  check_error( hipDeviceGetAttribute(&AsyncEngineCount, hipDeviceAttributeAsyncEngineCount, devID), HIPBLAS_STATUS_INTERNAL_ERROR);
  bool DO_INLINE_BOUT = AsyncEngineCount > 1;
  //*/
  
  check_error( hipMalloc( (void**)&d_A, (Am*An)*sizeof(T) ), HIPBLAS_STATUS_INTERNAL_ERROR);
  check_error( hipMalloc( (void**)&d_B, (Bm*Bn)*sizeof(T) ), HIPBLAS_STATUS_INTERNAL_ERROR);

  //setup streams
  hipStream_t inStream, outStream;
  check_error( hipStreamCreateWithFlags( &inStream, hipStreamNonBlocking), HIPBLAS_STATUS_INTERNAL_ERROR );
  if(DO_INLINE_BOUT)
    check_error( hipStreamCreateWithFlags( &outStream, hipStreamNonBlocking), HIPBLAS_STATUS_INTERNAL_ERROR );
  
  //call cpu API trmm
  check_error( 
    (status = kblasXtrsm(handle, inStream, outStream,
                         side, uplo, trans,diag,
                         m, n,
                         alpha, h_A, incA, d_A,
                                h_B, incB, d_B,
                         false, DO_INLINE_BOUT, false)
    ), status);
  //sync streams
  if(DO_INLINE_BOUT){
    check_error( hipStreamSynchronize( outStream ), HIPBLAS_STATUS_INTERNAL_ERROR);
  }else{
    hipStream_t compStream;
    check_error( hipblasGetStream(handle, &compStream), HIPBLAS_STATUS_INTERNAL_ERROR);
    check_error( hipStreamSynchronize( compStream ), HIPBLAS_STATUS_INTERNAL_ERROR);
    check_error( status = hipblasGetMatrixAsync( m, n, sizeof(T), d_B, incB, h_B, incB, inStream), status);
  }
  //revoke streams
  check_error( hipStreamDestroy( inStream ), HIPBLAS_STATUS_INTERNAL_ERROR);
  if(DO_INLINE_BOUT)
    check_error( hipStreamDestroy( outStream ), HIPBLAS_STATUS_INTERNAL_ERROR);

  /*check_error( hipHostUnregister( (void*)h_A ), HIPBLAS_STATUS_INTERNAL_ERROR );
  check_error( hipHostUnregister( (void*)h_B ), HIPBLAS_STATUS_INTERNAL_ERROR );*/

  //free device memory
  check_error( hipFree( d_A ), HIPBLAS_STATUS_INTERNAL_ERROR );
  check_error( hipFree( d_B ), HIPBLAS_STATUS_INTERNAL_ERROR );  
  check_error( hipGetLastError(), HIPBLAS_STATUS_EXECUTION_FAILED );
  return HIPBLAS_STATUS_SUCCESS;
}
//==============================================================================================
/*extern "C" {
  int kblas_strsm_async(
                        char side, char uplo, char trans, char diag,
                        int m, int n,
                        float alpha, const float *A, int incA,
                        float *B, int incB,
                        hipStream_t    stream){

    check_error(cublasSetKernelStream(stream));
    return kblasXtrsm(
                      side, uplo, trans, diag,
                      m, n,
                      alpha, A, incA,
                      B, incB);
  }
  int kblas_dtrsm_async(
                        char side, char uplo, char trans, char diag,
                        int m, int n,
                        double alpha, const double *A, int incA,
                        double *B, int incB,
                        hipStream_t    stream){

    check_error(cublasSetKernelStream(stream));
    return kblasXtrsm(
                      side, uplo, trans, diag,
                      m, n,
                      alpha, A, incA,
                      B, incB);
  }
  int kblas_ctrsm_async(
                        char side, char uplo, char trans, char diag,
                        int m, int n,
                        hipComplex alpha, const hipComplex *A, int incA,
                        hipComplex *B, int incB,
                        hipStream_t    stream){

    check_error(cublasSetKernelStream(stream));
    return kblasXtrsm(
                      side, uplo, trans, diag,
                      m, n,
                      alpha, A, incA,
                      B, incB);
  }
  int kblas_ztrsm_async(
                        char side, char uplo, char trans, char diag,
                        int m, int n,
                        hipDoubleComplex alpha, const hipDoubleComplex *A, int incA,
                        hipDoubleComplex *B, int incB,
                        hipStream_t    stream){

    check_error(cublasSetKernelStream(stream));
    return kblasXtrsm(
                      side, uplo, trans, diag,
                      m, n,
                      alpha, A, incA,
                      B, incB);
  }

  int kblas_strsm(
                  char side, char uplo, char trans, char diag,
                  int m, int n,
                  float alpha, const float *A, int incA,
                  float *B, int incB){
    return kblasXtrsm(
                      side, uplo, trans, diag,
                      m, n,
                      alpha, A, incA,
                      B, incB);
  }
  int kblas_dtrsm(
                  char side, char uplo, char trans, char diag,
                  int m, int n,
                  double alpha, const double *A, int incA,
                  double *B, int incB){
    return kblasXtrsm(
                      side, uplo, trans, diag,
                      m, n,
                      alpha, A, incA,
                      B, incB);
  }
  int kblas_ctrsm(
                  char side, char uplo, char trans, char diag,
                  int m, int n,
                  hipComplex alpha, const hipComplex *A, int incA,
                  hipComplex *B, int incB){
    return kblasXtrsm(
                      side, uplo, trans, diag,
                      m, n,
                      alpha, A, incA,
                      B, incB);
  }
  int kblas_ztrsm(
                  char side, char uplo, char trans, char diag,
                  int m, int n,
                  hipDoubleComplex alpha, const hipDoubleComplex *A, int incA,
                  hipDoubleComplex *B, int incB){
    return kblasXtrsm(
                      side, uplo, trans, diag,
                      m, n,
                      alpha, A, incA,
                      B, incB);
  }

}*/

//==============================================================================================

#define kblasXtrsm_async_BODY {                                                                          \
                                                                                                         \
  hipblasHandle_t cublas_handle;                                                                          \
  check_error( hipblasCreate(&cublas_handle), void() );                                                      \
  if( hipblasSetStream(cublas_handle, stream) != HIPBLAS_STATUS_SUCCESS ){                              \
    check_error( hipblasDestroy(cublas_handle), void());                                                  \
    return;                                                                                              \
  }                                                                                                      \
  hipblasSideMode_t  side_v2  = (side  == KBLAS_Left  ? HIPBLAS_SIDE_LEFT : HIPBLAS_SIDE_RIGHT);            \
  hipblasFillMode_t  uplo_v2  = (uplo  == KBLAS_Lower ? HIPBLAS_FILL_MODE_LOWER : HIPBLAS_FILL_MODE_UPPER); \
  hipblasOperation_t trans_v2 = (trans == KBLAS_Trans ? HIPBLAS_OP_T : HIPBLAS_OP_N);                       \
  hipblasDiagType_t  diag_v2  = (diag  == KBLAS_Unit  ? HIPBLAS_DIAG_UNIT : HIPBLAS_DIAG_NON_UNIT);         \
                                                                                                         \
  check_error( kblasXtrsm(cublas_handle,                                                                              \
                          side_v2, uplo_v2, trans_v2, diag_v2,                                                        \
                          m, n,                                                                                       \
                          &alpha, A, lda,                                                                             \
                                  B, ldb), void());                                                                             \
                                                                                                         \
  check_error( hipblasDestroy(cublas_handle), void());                                                    \
}
extern "C"{
void kblasStrsm_async(char side, char uplo, char trans, char diag,
                      int m, int n,
                      float alpha, const float *A, int lda,
                                         float *B, int ldb,
                      hipStream_t stream){
  kblasXtrsm_async_BODY
}

void kblasDtrsm_async(char side, char uplo, char trans, char diag,
                      int m, int n,
                      double alpha, const double *A, int lda,
                                          double *B, int ldb,
                      hipStream_t stream){
  kblasXtrsm_async_BODY
}
void kblasCtrsm_async(char side, char uplo, char trans, char diag,
                      int m, int n,
                      hipComplex alpha, const hipComplex *A, int lda,
                                             hipComplex *B, int ldb,
                      hipStream_t stream){
  kblasXtrsm_async_BODY
}
void kblasZtrsm_async(char side, char uplo, char trans, char diag,
                      int m, int n,
                      hipDoubleComplex alpha, const hipDoubleComplex *A, int lda,
                                                   hipDoubleComplex *B, int ldb,
                      hipStream_t stream){
  kblasXtrsm_async_BODY
}
}
//==============================================================================================

void kblasStrsm(char side, char uplo, char trans, char diag,
                int m, int n,
                float alpha, const float *A, int lda,
                                   float *B, int ldb){

  kblasStrsm_async(side, uplo, trans, diag,
                   m, n,
                   alpha, A, lda,
                          B, ldb,
                   0);
}

void kblasDtrsm(char side, char uplo, char trans, char diag,
                int m, int n,
                double alpha, const double *A, int lda,
                                    double *B, int ldb){

  kblasDtrsm_async(side, uplo, trans, diag,
                   m, n,
                   alpha, A, lda,
                          B, ldb,
                   0);
}
void kblasCtrsm(char side, char uplo, char trans, char diag,
                int m, int n,
                hipComplex alpha, const hipComplex *A, int lda,
                                       hipComplex *B, int ldb){

  kblasCtrsm_async(side, uplo, trans, diag,
                   m, n,
                   alpha, A, lda,
                          B, ldb,
                   0);

}
void kblasZtrsm(char side, char uplo, char trans, char diag,
                int m, int n,
                hipDoubleComplex alpha, const hipDoubleComplex *A, int lda,
                                             hipDoubleComplex *B, int ldb){

  kblasZtrsm_async(side, uplo, trans, diag,
                   m, n,
                   alpha, A, lda,
                          B, ldb,
                   0);
}

//==============================================================================================

hipblasStatus_t kblasStrsm(hipblasHandle_t handle,
                          hipblasSideMode_t side, hipblasFillMode_t uplo,
                          hipblasOperation_t trans, hipblasDiagType_t diag,
                          int m, int n,
                          const float *alpha,
                          const float *A, int lda,
                                float *B, int ldb){
  return kblasXtrsm(handle,
                    side, uplo, trans, diag,
                    m, n,
                    alpha, A, lda,
                           B, ldb);
}
hipblasStatus_t kblasDtrsm(hipblasHandle_t handle,
                          hipblasSideMode_t side, hipblasFillMode_t uplo,
                          hipblasOperation_t trans, hipblasDiagType_t diag,
                          int m, int n,
                          const double *alpha,
                          const double *A, int lda,
                                double *B, int ldb){
  return kblasXtrsm(handle,
                    side, uplo, trans, diag,
                    m, n,
                    alpha, A, lda,
                           B, ldb);
}
hipblasStatus_t kblasCtrsm(hipblasHandle_t handle,
                          hipblasSideMode_t side, hipblasFillMode_t uplo,
                          hipblasOperation_t trans, hipblasDiagType_t diag,
                          int m, int n,
                          const hipComplex *alpha,
                          const hipComplex *A, int lda,
                                hipComplex *B, int ldb){
  return kblasXtrsm(handle,
                    side, uplo, trans, diag,
                    m, n,
                    alpha, A, lda,
                           B, ldb);
}
hipblasStatus_t kblasZtrsm(hipblasHandle_t handle,
                          hipblasSideMode_t side, hipblasFillMode_t uplo,
                          hipblasOperation_t trans, hipblasDiagType_t diag,
                          int m, int n,
                          const hipDoubleComplex *alpha,
                          const hipDoubleComplex *A, int lda,
                                hipDoubleComplex *B, int ldb){
  return kblasXtrsm(handle,
                    side, uplo, trans, diag,
                    m, n,
                    alpha, A, lda,
                           B, ldb);
}
//==============================================================================================

hipblasStatus_t kblas_strsm(hipblasHandle_t handle,
                           hipblasSideMode_t side, hipblasFillMode_t uplo,
                           hipblasOperation_t trans, hipblasDiagType_t diag,
                           int m, int n,
                           const float *alpha,
                           const float *A, int lda,
                                 float *B, int ldb){
  return kblasXtrsm_cpu(handle,
                        side, uplo, trans, diag,
                        m, n,
                        alpha, A, lda,
                              B, ldb);
}
hipblasStatus_t kblas_dtrsm(hipblasHandle_t handle,
                           hipblasSideMode_t side, hipblasFillMode_t uplo,
                           hipblasOperation_t trans, hipblasDiagType_t diag,
                           int m, int n,
                           const double *alpha,
                           const double *A, int lda,
                                 double *B, int ldb){
  return kblasXtrsm_cpu(handle,
                        side, uplo, trans, diag,
                        m, n,
                        alpha, A, lda,
                              B, ldb);
}
hipblasStatus_t kblas_ctrsm(hipblasHandle_t handle,
                           hipblasSideMode_t side, hipblasFillMode_t uplo,
                           hipblasOperation_t trans, hipblasDiagType_t diag,
                           int m, int n,
                           const hipComplex *alpha,
                           const hipComplex *A, int lda,
                                 hipComplex *B, int ldb){
  return kblasXtrsm_cpu(handle,
                        side, uplo, trans, diag,
                        m, n,
                        alpha, A, lda,
                              B, ldb);
}
hipblasStatus_t kblas_ztrsm(hipblasHandle_t handle,
                           hipblasSideMode_t side, hipblasFillMode_t uplo,
                           hipblasOperation_t trans, hipblasDiagType_t diag,
                           int m, int n,
                           const hipDoubleComplex *alpha,
                           const hipDoubleComplex *A, int lda,
                                 hipDoubleComplex *B, int ldb){
  return kblasXtrsm_cpu(handle,
                        side, uplo, trans, diag,
                        m, n,
                        alpha, A, lda,
                              B, ldb);
}





